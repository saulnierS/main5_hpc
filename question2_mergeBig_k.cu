#include "hip/hip_runtime.h"
//%%writefile question2.cu
#include <stdio.h> 
#include <stdlib.h>
#include "fonctionsCPU.h"
#define N 67107840//taille max du tableau =d dans le projet
//#define N 10000
#define threadsPerBlock 1024
#define numBlock 65535
//*****************************************************************************
//Fonctions GPU (merge tableau)
//*****************************************************************************

__device__ void pathBig_k(int *A, int *B, int *Path, int size_A, int size_B, int size_M)
{
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i<size_M; i = i+blockDim.x*gridDim.x)
    {
      int K[2],P[2],Q[2];
      int offset;

      //if (i==0) printf("A: %d B:%d M:%d\n", size_A, size_B, size_M);
      if (i>size_A)
      {
          K[0]=i-size_A;
          K[1]=size_A;
          P[0]=size_A;
          P[1]=i-size_A;
      }
      else
      {
          K[0]=0;
          K[1]=i;
          P[0]=i;
          P[1]=0;
          //if (i==0) printf("K(%d,%d) et P(%d,%d)\n",K[0],K[1],P[0],P[1]);
      }
      while (1)
      {
          offset=abs(K[1]-P[1])/2;
          //if (i==0) printf("K(%d,%d) et P(%d,%d)\n",K[0],K[1],P[0],P[1]);
          Q[1]=K[1]-offset;
          Q[0]=K[0]+offset;
          if (Q[1] >= 0 && Q[0] <= size_B && (Q[1]== size_A || Q[0]==0 || A[Q[1]]>B[Q[0]-1]))//verif
          {
              if (Q[0]==size_B || Q[1]==0 || A[Q[1]-1]<=B[Q[0]])//verif
              {
                  if (Q[1]<size_A && (Q[0]==size_B || A[Q[1]]<=B[Q[0]]))//verif
                  {
                      Path[i]=1;//=on suit A (lignes verticales)
                      Path[i+size_M]=Q[1];
                  }
                  else
                  {
                      Path[i]=0;//=on suit B (lignes horizontales)
                      Path[i+size_M]=Q[0];
                  }
                  break;
              }
              else
              {
                  K[0]=Q[0]+1;
                  K[1]=Q[1]-1;
              }
          }
          else
          {
            P[0]=Q[0]-1;
            P[1]=Q[1]+1;
          }
      }
    }
}


__device__ void mergeBig_k(int *A, int *B, int *M,int *Path, int size_A, int size_B, int size_M)
{
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i<size_M; i = i+blockDim.x*gridDim.x)
    {
        if (Path[i]==1)
          M[i]=A[Path[i+size_M]];
        else if (Path[i]==0)
          M[i]=B[Path[i+size_M]];
        else
          printf("ERROR thread num %d block %d",i,blockIdx.x);
                  
    }
}

__global__ void sortManager(int *A, int *B, int *M, int *Path, int size_A, int size_B, int size_M)
{

    pathBig_k(A, B, Path, size_A, size_B, size_M);
    mergeBig_k(A, B, M, Path, size_A, size_B, size_M);
}
 
//*****************************************************************************
//MAIN
//*****************************************************************************
int main() {
  srand (time (NULL));


  /*Déclaration des variables CPU*/

  /*Taille des tableaux*/
  int h_taille_A=rand()%(N-1)+1;//j ai rajouter 1 comme ca on peut pas piocher 0
  int h_taille_B=N-h_taille_A;//pour eviter d avoir 0 si on a piocher 10 normalement on ne devrait pas piocher 11
  int h_taille_M=h_taille_A+h_taille_B; //en fait je pense que c est plus le nombre de threads 
  
  printf("taille alea A:%d, B:%d N:%d",h_taille_A,h_taille_B,N);
  if (h_taille_A < h_taille_B)
  {
      int tpm=h_taille_A;
      h_taille_A=h_taille_B;
      h_taille_B=tpm;
  }

  /*Partie test*/
  /*int h_taille_A=9;
  int h_taille_B=7;
  int h_taille_M=16;
  */

  /*Tableaux et allocation memoire*/
  int *h_A;
  int *h_B;
  int *h_M;
  int *h_Path;
  h_A=(int *)malloc(h_taille_A*sizeof(int));
  h_B=(int *)malloc(h_taille_B*sizeof(int));
  h_M=(int *)malloc(h_taille_M*sizeof(int));
  h_Path=(int *)malloc(2*h_taille_M*sizeof(int));
 

  /*Déclaration des variables GPU*/ 
    int *d_A; 
    int *d_B; 
    int *d_M;
    int *d_Path;
    hipMalloc(&d_A,h_taille_A*sizeof(int)); 
    hipMalloc(&d_B,h_taille_B*sizeof(int));
    hipMalloc(&d_M,h_taille_M*sizeof(int));
    hipMalloc(&d_Path,2*h_taille_M*sizeof(int));
    

   
  /*Initialisation et preparation des tableaux*/
  for (int i=0; i<h_taille_A;i++)
  {
    h_A[i]=rand()%10000;
  }
  for (int i=0; i<h_taille_B;i++)
  {
    h_B[i]=rand()%10000;
  }
  
  tri_fusion(h_A, h_taille_A);
  tri_fusion(h_B, h_taille_B);


  //test(h_A, h_B);


  printf("\n");
  /*Affichage*/
  printf("***A***\n");
  for (int i=0; i<10; i=i+1)
    printf("%d\n",h_A[i]);
  printf("***B***\n");
  for (int i=0; i<10; i=i+1)
    printf("%d\n",h_B[i]);


  /*Transfert la mémoire du cpu vers le gpu*/
  hipMemcpy(d_A, h_A, h_taille_A*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, h_taille_B*sizeof(int), hipMemcpyHostToDevice);
  //hipMemcpy(d_M, h_M, h_taille_M*sizeof(int), hipMemcpyHostToDevice);


  /*Merge tableau*/
  sortManager<<<numBlock,threadsPerBlock>>>(d_A, d_B, d_M, d_Path, h_taille_A, h_taille_B, h_taille_M);

  /*Transfert la mémoire du gpu vers le cpu*/
  hipMemcpy(h_M, d_M, h_taille_M*sizeof(int), hipMemcpyDeviceToHost);

  /*Affichage du resultat*/
  printf("***M***\n");
  for (int i=0; i<10; i=i+1)
    printf("%d\n",h_M[i]);
  
  if (verif_trie(h_M,h_taille_M)==1)
    printf("\nok tableau trie\n");
  else
    printf("\nKO probleme a l indice %d\n",verif_trie(h_M,h_taille_M));

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_M);
  hipFree(d_Path);
  free(h_A);
  free(h_B);
  free(h_M);
  free(h_Path);
 

    return 0;
} 