#include "hip/hip_runtime.h"
//%%writefile question4.cu
#include <stdio.h> 
#include <stdlib.h>
#include "fonctionsCPU.h"
#define N 67107840//taille max du tableau =d dans le projet
#define threadsPerBlock 1024
#define numBlock 65535
//*****************************************************************************
//Fonctions GPU (merge tableau)
//*****************************************************************************

__global__ void pathBig_k(int *A, int *B, int *Path, int size_A, int size_B, int size_M)
{
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i<size_M; i = i+blockDim.x*gridDim.x)
    {
      int K[2],P[2],Q[2];
      int offset;

      if (i==0) printf("A: %d B:%d M:%d\n", size_A, size_B, size_M);
      if (i>size_A)
      {
          K[0]=i-size_A;
          K[1]=size_A;
          P[0]=size_A;
          P[1]=i-size_A;
      }
      else
      {
          K[0]=0;
          K[1]=i;
          P[0]=i;
          P[1]=0;
          if (i==0) printf("K(%d,%d) et P(%d,%d)\n",K[0],K[1],P[0],P[1]);
      }
      while (1)
      {
          offset=(K[1]-P[1])/2;
          //if (i==0) printf("K(%d,%d) et P(%d,%d)\n",K[0],K[1],P[0],P[1]);
          Q[1]=K[1]-offset;
          Q[0]=K[0]+offset;
          if (Q[1] >= 0 && Q[0] <= size_B && (Q[1]== size_A || Q[0]==0 || A[Q[1]]>B[Q[0]-1]))//verif
          {
              if (Q[0]==size_B || Q[1]==0 || A[Q[1]-1]<=B[Q[0]])//verif
              {
                  if (Q[1]<size_A && (Q[0]==size_B || A[Q[1]]<=B[Q[0]]))//verif
                  {
                      Path[i]=1;//=on suit A (lignes verticales)
                      Path[i+size_M]=Q[1];
                  }
                  else
                  {
                      Path[i]=0;//=on suit B (lignes horizontales)
                      Path[i+size_M]=Q[0];
                  }
                  break;
              }
              else
              {
                  K[0]=Q[0]+1;
                  K[1]=Q[1]-1;
              }
          }
          else
          {
            P[0]=Q[0]-1;
            P[1]=Q[1]+1;
          }
      }
    }
}


__global__ void mergeBig_k(int *A, int *B, int *M,int *Path, int size_A, int size_B, int size_M)
{
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i<size_M; i = i+blockDim.x*gridDim.x)
    {
        if (Path[i]==1)
          M[i]=A[Path[i+size_M]];
        else if (Path[i]==0)
          M[i]=B[Path[i+size_M]];
        else
          printf("ERROR thread num %d block %d",i,blockIdx.x);
                  
    }
}
//*****************************************************************************
//MAIN
//*****************************************************************************
int main() {
    srand (time (NULL));
    /*Déclaration des variables CPU*/
    /*Partie test*/
    
      int h_nb_tab=3000;

      int h_taille_A=9;
      int h_taille_B=7; 
      int h_taille_M=16;
      int h_taille_M_total=h_nb_tab*(h_taille_A+h_taille_B);   
  

    /*Tableaux et allocation memoire*/
    int **h_A;
    int **h_B;
    int **h_M;
    int **h_Path;
    int *h_M_total;

    /*Alloction memoire*/
    h_A = (int **) malloc( h_nb_tab* sizeof(int *) );
    h_B = (int **) malloc( h_nb_tab* sizeof(int *) );
    h_M = (int **) malloc( h_nb_tab* sizeof(int *) );
    h_Path = (int **) malloc( h_nb_tab* sizeof(int *) );
    h_M_total = (int *) malloc(h_taille_M_total* sizeof(int));

    /*Choix aleatoire des tailles des tableaux*/
    for (int i=0; i<h_nb_tab; i=i+1)
    {
        /*Alloction memoire*/
        h_A[i]=(int *)malloc(h_taille_A*sizeof(int));
        h_B[i]=(int *)malloc(h_taille_B*sizeof(int)); 
        h_M[i]=(int *)malloc(h_taille_M*sizeof(int)); 
        h_Path[i]=(int *)malloc(h_taille_M*sizeof(int));

    }
  

    /*Initialisation et preparation des tableaux*/
    for (int i=0; i<h_nb_tab;i++)
    {
        for(int j=0; j<h_taille_A;j++)
          h_A[i][j]=rand()%10000;
        for(int j=0; j<h_taille_B;j++)
          h_B[i][j]=rand()%10000;
    }

    /*Trie des tableaux*/
    for (int i=0; i<h_nb_tab;i++)
    {
        tri_fusion(h_A[i], h_taille_A);
        tri_fusion(h_B[i], h_taille_B);
    }

    /*Affichage*/
    
    printf("\n***A***\n");
    for (int i=0; i<h_nb_tab;i++)
    {
        printf("\ntaille du tableau %d : %d\n",i,h_taille_A);
        for(int j=0; j<h_taille_A;j++)
          printf("%d,",h_A[i][j]);
    }
    printf("\n***B***\n");
    for (int i=0; i<h_nb_tab;i++)
    {
        printf("\ntaille du tableau %d : %d\n",i,h_taille_B);
        for(int j=0; j<h_taille_B;j++)
          printf("%d,",h_B[i][j]);
    }
    
    /*Declaration variable GPU*/
    int *d_A; 
    int *d_B; 
    int *d_M;
    int *d_Path;
    int *d_M_total;
    //hipStream_t stream;

    /*Alloction memoire*/
    hipMalloc(&d_A,h_taille_A*sizeof(int));
    hipMalloc(&d_B,h_taille_B*sizeof(int));
    hipMalloc(&d_M,h_taille_M*sizeof(int));
    hipMalloc(&d_Path,h_taille_M*sizeof(int));
    hipMalloc(&d_M_total,h_taille_M_total*sizeof(int));

    
    for (int i=0; i<h_nb_tab;i++)
    {
        /*Transfert la mémoire du cpu vers le gpu*/
        hipMemcpy(d_A, h_A[i], h_taille_A*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B[i], h_taille_B*sizeof(int), hipMemcpyHostToDevice);
        /*Merge tableau*/
        pathBig_k<<<numBlock,threadsPerBlock>>>(d_A, d_B, d_Path, h_taille_A, h_taille_B, h_taille_M);
        hipDeviceSynchronize();
        mergeBig_k<<<numBlock,threadsPerBlock>>>(d_A, d_B, d_M, d_Path, h_taille_A, h_taille_B, h_taille_M);
        /*Transfert la mémoire du gpu vers le cpu*/
        hipMemcpy(h_M[i], d_M, h_taille_M*sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }
    int h_taille_tpm=0;
    int *h_tpm;
    int *d_tpm;
    int *d_Path_tpm;
    for (int j=0; j<h_taille_M; j++)
            h_M_total[j]=h_M[0][j];
    for (int i=0;i<h_nb_tab-1; i=i+1)
    {
        /*Preparation de la memoire*/
        h_taille_tpm+=h_taille_M;
        h_tpm=(int *)malloc(h_taille_tpm*sizeof(int));
        hipMalloc(&d_tpm,h_taille_tpm*sizeof(int));
        hipMalloc(&d_Path_tpm,(h_taille_M+h_taille_tpm)*sizeof(int));
        for (int j=0; j<h_taille_tpm; j++)
            h_tpm[j]=h_M_total[j];
        
        
        /*Transfert la memoire du cpu vers le gpu*/
        hipMemcpy(d_tpm, h_tpm, h_taille_tpm*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_M, h_M[i+1], h_taille_M*sizeof(int), hipMemcpyHostToDevice);
        
        
        /*Merge tableau*/
        pathBig_k<<<numBlock,threadsPerBlock>>>(d_tpm, d_M, d_Path_tpm, h_taille_tpm, h_taille_M, h_taille_tpm+h_taille_M);
        hipDeviceSynchronize();
        mergeBig_k<<<numBlock,threadsPerBlock>>>(d_tpm, d_M, d_M_total, d_Path_tpm, h_taille_tpm, h_taille_M, h_taille_tpm+h_taille_M);
        
        
        /*Transfert la mémoire du gpu vers le cpu*/
        hipMemcpy(h_M_total, d_M_total, (h_taille_tpm+h_taille_M)*sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipFree(d_tpm);
        hipFree(d_Path_tpm);
        free(h_tpm);
        
    }
    
    

    /*Affichage du resultat*/
    printf("\n***M***\n");
    for (int i=0; i<h_nb_tab; i++)
    {
        for (int j=0;j<h_taille_M;j++)
          printf("%d,",h_M[i][j]);
        printf("\n");
    }
    printf("\n***M tot***\n");
    for (int i=0; i<h_taille_M_total; i++)
    {
          printf("%d,",h_M_total[i]);
    }

    if (verif_trie(h_M_total,h_taille_M_total)==1)
      printf("\nok tableau trie\n");
    else
      printf("\nKO recommencer\n");

    /*Liberation de la memoire GPU*/
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_M);
    hipFree(d_Path);
    hipFree(d_M_total);


    /*Liberation de la memoire CPU*/
    for(int i=0; i<h_nb_tab;i++)
    {
      free(h_A[i]);
      free(h_B[i]);
      free(h_M[i]);
      free(h_Path[i]);
    }
    free(h_A);
    free(h_B);
    free(h_M);
    free(h_Path);
    free(h_M_total);
    

   
  

    return 0;
} 