#include "hip/hip_runtime.h"
//%%writefile question3.cu
#include <stdio.h> 
#include <stdlib.h>
#include "fonctionsCPU.h"
#define N 100000 //taille max du tableau =d dans le projet
#define threadsPerBlock 1024
#define numBlock 65535
//*****************************************************************************
//Fonctions GPU (merge tableau)
//*****************************************************************************

__device__ void pathBig_k(int *A, int *B, int *Path, int size_A, int size_B, int size_M)
{
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i<size_M; i = i+blockDim.x*gridDim.x)
    {
      int K[2],P[2],Q[2];
      int offset;

      //if (i==0) printf("A: %d B:%d M:%d\n", size_A, size_B, size_M);
      if (i>size_A)
      {
          K[0]=i-size_A;
          K[1]=size_A;
          P[0]=size_A;
          P[1]=i-size_A;
      }
      else
      {
          K[0]=0;
          K[1]=i;
          P[0]=i;
          P[1]=0;
          //if (i==0) printf("K(%d,%d) et P(%d,%d)\n",K[0],K[1],P[0],P[1]);
      }
      while (1)
      {
          offset=abs(K[1]-P[1])/2;
          //if (i==0) printf("K(%d,%d) et P(%d,%d)\n",K[0],K[1],P[0],P[1]);
          Q[1]=K[1]-offset;
          Q[0]=K[0]+offset;
          if (Q[1] >= 0 && Q[0] <= size_B && (Q[1]== size_A || Q[0]==0 || A[Q[1]]>B[Q[0]-1]))//verif
          {
              if (Q[0]==size_B || Q[1]==0 || A[Q[1]-1]<=B[Q[0]])//verif
              {
                  if (Q[1]<size_A && (Q[0]==size_B || A[Q[1]]<=B[Q[0]]))//verif
                  {
                      Path[i]=1;//=on suit A (lignes verticales)
                      Path[i+size_M]=Q[1];
                  }
                  else
                  {
                      Path[i]=0;//=on suit B (lignes horizontales)
                      Path[i+size_M]=Q[0];
                  }
                  break;
              }
              else
              {
                  K[0]=Q[0]+1;
                  K[1]=Q[1]-1;
              }
          }
          else
          {
            P[0]=Q[0]-1;
            P[1]=Q[1]+1;
          }
      }
    }
}


__device__ void mergeBig_k(int *A, int *B, int *M,int *Path, int size_A, int size_B, int size_M)
{
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i<size_M; i = i+blockDim.x*gridDim.x)
    {
        if (Path[i]==1)
          M[i]=A[Path[i+size_M]];
        else if (Path[i]==0)
          M[i]=B[Path[i+size_M]];
        else
          printf("ERROR thread num %d block %d",i,blockIdx.x);
                  
    }
}

__global__ void sortManager_GPU(int *A, int *B, int *M,int *Path, int size_A, int size_B, int size_M)
{
    pathBig_k(A, B, Path, size_A, size_B, size_M);
    mergeBig_k(A, B, M, Path, size_A, size_B, size_M);
}

void sortManager_CPU(int *h_M,int h_size_A,int h_size_B,int h_slice_size,int i)
{
    
    //printf("\n***slice number***  %d\n", i);
    //printf("\n***slice size***  %d\n", h_slice_size);
    /*Variables CPU*/ 
    int h_size_M_tmp= h_size_A+h_size_B;
    int *h_A;
    int *h_B;
    int *h_M_tmp;
    h_A=(int *)malloc(h_size_A*sizeof(int));
    h_B=(int *)malloc(h_size_B*sizeof(int));
    h_M_tmp=(int *)malloc(h_size_M_tmp*sizeof(int));

    /*Remplir A et B*/
    for (int j=0; j<h_size_A; j++)
    {
        h_A[j] = h_M[i*h_slice_size+j];
        //printf("\nindice A: %d indice M: %d",j,i*h_slice_size+j );
    }
        
    for (int j=0; j<h_size_B; j++)
    {
         h_B[j] = h_M[i*h_slice_size+j+h_size_A];
         //printf("\nindice B: %d indice M: %d",j,i*h_slice_size+h_size_A );
    }
    /* 
    printf("\nA\n");
    for (int j=0; j<h_size_A; j++)
        printf(" %d ", h_A[j]);
    printf("\nB\n");
    for (int j=0; j<h_size_B; j++)
        printf(" %d ", h_B[j]);
    printf("\n");
    */
 
    /*Variables GPU*/
    int *d_A;
    int *d_B;
    int *d_M_tmp;
    int *d_Path_tmp;
    hipMalloc(&d_A,h_size_A*sizeof(int));
    hipMalloc(&d_B,h_size_B*sizeof(int));
    hipMalloc(&d_M_tmp,h_size_M_tmp*sizeof(int));
    hipMalloc(&d_Path_tmp,h_size_M_tmp*sizeof(int));

  
    /*Transfert*/
    hipMemcpy(d_A, h_A,h_size_A*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B,h_size_B*sizeof(int), hipMemcpyHostToDevice);
 
    /*Kernel*/
    if (h_size_A<h_size_B)
    {

        sortManager_GPU<<<numBlock,threadsPerBlock>>>(d_B, d_A, d_M_tmp, d_Path_tmp, h_size_B, h_size_A, h_size_M_tmp);
        hipDeviceSynchronize();
    }
    else
    {

        sortManager_GPU<<<numBlock,threadsPerBlock>>>(d_A, d_B, d_M_tmp, d_Path_tmp, h_size_A, h_size_B, h_size_M_tmp);
        hipDeviceSynchronize();    
    }
    
    /*Transfert memoire GPU*/
    hipMemcpy(h_M_tmp, d_M_tmp, h_size_M_tmp*sizeof(int), hipMemcpyDeviceToHost);

    /*Affichage du resultat*/
     /*     printf("\nMerge\n");
          for (int k=0; k<h_size_M_tmp; k=k+1)
            printf("%d , indice[%d]\n",h_M_tmp[k],k);
          if (verif_trie(h_M_tmp,h_size_M_tmp)==1)
            printf("ok tableau trie");
          else
            printf("KO tmp recommencer %d \n",verif_trie(h_M_tmp,h_size_M_tmp) );
     */
    /*Copie de h_M_tmp dans h_M*/
    //printf("\n***M***\n");
    for (int j=0; j<h_size_M_tmp; j++)
    {
        h_M[i*h_slice_size+j]=h_M_tmp[j];
        //printf("[%d] %d ,",i*h_slice_size+j,h_M_tmp[j] );
    }
    //printf("\n***************************\n");
    
    
    /*Liberation*/
    free(h_A);
    free(h_B);
    free(h_M_tmp);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_M_tmp);
    hipFree(d_Path_tmp);
}
//*****************************************************************************
//MAIN
//*****************************************************************************
int main() {
  srand (time (NULL));


  /*Déclaration des variables CPU*/
  /*Taille des tableaux*/
  //int h_taille_M=rand()%(N-1)+1;  
  //int h_taille_M=14; 
  int h_taille_M=N; 

  printf("Taille de M : %d\n",h_taille_M);

  /*Tableaux et allocation memoire*/
  int *h_M;
  h_M=(int *)malloc(h_taille_M*sizeof(int));
 
  /*Déclaration des variables GPU*/  
  int *d_M;
  hipMalloc(&d_M,h_taille_M*sizeof(int));

  /*Initialisation et preparation des tableaux*/
  for (int i=0; i<h_taille_M;i++)
    h_M[i]=rand()%10000;

  //printf("***M***\n");
  //for (int i=0; i<h_taille_M; i=i+1)
    //printf("%d\n",h_M[i]);

  /*Merge tableau*/
  int h_slice_size=1;
  int h_slice_number=h_taille_M/2;
  int h_slice_reste_precedent=0;
  int h_slice_reste=0;
  while (h_slice_number > 0)
  {   
      /*Mise a jour taille et indices*/
      h_slice_size=2*h_slice_size;
      
      h_slice_reste_precedent=h_slice_reste;
      h_slice_reste=h_taille_M%h_slice_size;
      h_slice_number=h_taille_M/h_slice_size;
      

      for (int i=0; i<h_slice_number; i++)
      {   
          sortManager_CPU(h_M,h_slice_size/2,h_slice_size/2,h_slice_size,i);
          
      }
      if (h_slice_reste_precedent!=0 && h_slice_reste!=0)
      {
              int h_taille_A=h_slice_reste-h_slice_reste_precedent;
              int h_taille_B=h_slice_reste_precedent;
              sortManager_CPU(h_M,h_taille_A,h_taille_B,h_slice_size,h_slice_number);

      }

      /*Affichage du resultat*/
      /*printf("***M***\n");
      for (int k=0; k<h_taille_M; k=k+1)
          printf("%d\n",h_M[k]);
      */

          
  }


  /*Affichage du resultat*/
  //printf("***M***\n");
  //for (int i=0; i<h_taille_M; i=i+1)
    //printf("%d\n",h_M[i]);
  if (verif_trie(h_M,h_taille_M)==1)
    printf("ok tableau trie");
  else
    printf("KO recommencer %d ",verif_trie(h_M,h_taille_M) );
  free(h_M);

 

    return 0;
}