#include "hip/hip_runtime.h"
//%%writefile question1_shared.cu
#include <stdio.h> 
#include <stdlib.h>
#include <string.h>
#include "fonctionsCPU.h"
#define N 1024//taille max du tableau =d dans le projet
#define threadsPerBlock 1024
//*****************************************************************************
//Fonctions GPU (merge tableau)
//*****************************************************************************

__device__ void mergeSmall_k(int *A, int *B, int *M, int size_A, int size_B, int size_M)
{


    
    //for(int i = blockIdx.x * blockDim.x + threadIdx.x; i<size_M; i = i+blockDim.x*gridDim.x)
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i< size_M)
    {
      
     
     /*Merge*/
      int K[2],P[2],Q[2];
      int offset;

      //if (i==0) printf("A: %d B:%d M:%d\n", size_A, size_B, size_M);
      if (i>size_A)
      {
          K[0]=i-size_A;
          K[1]=size_A;
          P[0]=size_A;
          P[1]=i-size_A;
      }
      else
      {
          K[0]=0;
          K[1]=i;
          P[0]=i;
          P[1]=0;
          //if (i==0) printf("K(%d,%d) et P(%d,%d)\n",K[0],K[1],P[0],P[1]);
      }
      while (1)
      {
          offset=abs(K[1]-P[1])/2;
          //if (i==0) printf("K(%d,%d) et P(%d,%d)\n",K[0],K[1],P[0],P[1]);
          Q[1]=K[1]-offset;
          Q[0]=K[0]+offset;
          if (Q[1] >= 0 && Q[0] <= size_B && (Q[1]== size_A || Q[0]==0 || A[Q[1]]>B[Q[0]-1]))//verif
          {
              if (Q[0]==size_B || Q[1]==0 || A[Q[1]-1]<=B[Q[0]])//verif
              {
                  if (Q[1]<size_A && (Q[0]==size_B || A[Q[1]]<=B[Q[0]]))//verif
                  {
                      M[i]=A[Q[1]];
                  }
                  else
                  {
                      M[i]=B[Q[0]]; 
                  }
                  break;
              }
              else
              {
                  K[0]=Q[0]+1;
                  K[1]=Q[1]-1;
              }
          }
          else
          {
            P[0]=Q[0]-1;
            P[1]=Q[1]+1;
          }
      }
    }
  //printf("pour %d on fait %d tours \n",blockIdx.x * blockDim.x + threadIdx.x,c);
}

__global__ void sortManager(int *A, int *B, int *M, int size_A, int size_B, int size_M)
{
    /*Chargement de A et B dans la shared memory*/
    /*Comme on a une seule shared memory*/
    __shared__ int shared_AB[N];
 
    int* s_A = (int*) &shared_AB[0];
    int* s_B = (int*) &s_A[size_A];
 
    __syncthreads();
 
    int i = blockDim.x * blockIdx.x + threadIdx.x;
 
    if (i < size_A)
        s_A[i] = A[i];
 
    if (i < size_B)
      s_B[i] = B[i];
 
    __syncthreads();

    mergeSmall_k(s_A, s_B, M, size_A, size_B, size_M);
}

 
//*****************************************************************************
//MAIN
//*****************************************************************************
int main(int argc, char *argv[]) {
    
  srand (time (NULL));
  int numThreads=threadsPerBlock;
  /*Déclaration des variables CPU*/
  /*Taille des tableaux*/

  int h_taille_A=rand()%(N-1)+1;//j ai rajouter 1 comme ca on peut pas piocher 0
  int h_taille_B=N-h_taille_A;//pour eviter d avoir 0 si on a piocher 10 normalement on ne devrait pas piocher 11
  int h_taille_M=h_taille_A+h_taille_B; //en fait je pense que c est plus le nombre de threads 

  for (int i=0; i<argc-1; i=i+1)
  {
      if (strcmp(argv[i],"--tailleA")==0 && atoi(argv[i+1])<N )
          h_taille_A=atoi(argv[i+1]);
      if (strcmp(argv[i],"--tailleB")==0 && atoi(argv[i+1])<N)
          h_taille_B=atoi(argv[i+1]);
      if (strcmp(argv[i],"--tailleA")==0 && atoi(argv[i+1])<threadsPerBlock )
          numThreads=atoi(argv[i+1]);
      //printf("%d %d\n",i,strcmp(argv[i],"--tailleA"));  
  }
  printf("taille alea A:%d, B:%d N:%d",h_taille_A,h_taille_B,N);
  if (h_taille_A < h_taille_B)
  {
      int tpm=h_taille_A;
      h_taille_A=h_taille_B;
      h_taille_B=tpm;
  }
  /*Partie test*/
  /*
  int h_taille_A=9;
  int h_taille_B=7;
  int h_taille_M=16;
  */

  /*Tableaux et allocation memoire*/
  int *h_A;
  int *h_B;
  int *h_M;
  h_A=(int *)malloc(h_taille_A*sizeof(int));
  h_B=(int *)malloc(h_taille_B*sizeof(int));
  h_M=(int *)malloc(h_taille_M*sizeof(int));
 

  /*Déclaration des variables GPU*/ 
  int *d_A; 
  int *d_B; 
  int *d_M;
  hipMalloc(&d_A,h_taille_A*sizeof(int)); 
  hipMalloc(&d_B,h_taille_B*sizeof(int));
  hipMalloc(&d_M,h_taille_M*sizeof(int));
    

   
  /*Initialisation et preparation des tableaux*/
  for (int i=0; i<h_taille_A;i++)
    h_A[i]=rand()%10000;
  
  for (int i=0; i<h_taille_B;i++)
    h_B[i]=rand()%10000;
  
  tri_fusion(h_A, h_taille_A);
  tri_fusion(h_B, h_taille_B);


  //test(h_A, h_B);


  /*Affichage*/
  printf("***A***\n");
  for (int i=0; i<10; i=i+1)
    printf("%d\n",h_A[i]);
  printf("***B***\n");
  for (int i=0; i<10; i=i+1)
    printf("%d\n",h_B[i]);


  /*Transfert la mémoire du cpu vers le gpu*/
  hipMemcpy(d_A, h_A, h_taille_A*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, h_taille_B*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_M, h_M, h_taille_M*sizeof(int), hipMemcpyHostToDevice);


  /*Merge tableau*/
  sortManager<<<1,numThreads>>>(d_A, d_B, d_M, h_taille_A, h_taille_B, h_taille_M);

  /*Transfert la mémoire du gpu vers le cpu*/
  hipMemcpy(h_M, d_M, h_taille_M*sizeof(int), hipMemcpyDeviceToHost);

  /*Affichage du resultat*/
  printf("***M***\n");
  for (int i=0; i<10; i=i+1)
    printf("%d\n",h_M[i]);

  if (verif_trie(h_M,h_taille_M)==1)
    printf("\n ok tableau trie\n");
    else
  {
    printf("\n KO probleme a l indice %d\n",verif_trie(h_M,h_taille_M));
    printf("%d %d %d %d\n",h_M[verif_trie(h_M,h_taille_M)-1],h_M[verif_trie(h_M,h_taille_M)], h_M[verif_trie(h_M,h_taille_M)+1], h_M[verif_trie(h_M,h_taille_M)+2] );
  }
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_M);
  free(h_A);
  free(h_B);
  free(h_M);
 

    return 0;
} 