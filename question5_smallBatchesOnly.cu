#include "hip/hip_runtime.h"
//%%writefile question5_smallBatchesOnly.cu
#include <stdio.h> 
#include <stdlib.h>
#include <string.h>
#include "fonctionsCPU.h"
#define N 1024
#define threadsPerBlock 1024
#define numBlock 65535
//*****************************************************************************
//Fonctions GPU (merge tableau)
//*****************************************************************************

__device__ void mergeSmallBatch_k(int *A, int *B, int *M, int size_A, int size_B, int size_M, int slice_size)
{

    int i = threadIdx.x;
    if (i < size_A+size_B)
    {
        //printf("\n blockIdx = %d, threadIdx = %d \n",blockIdx.x,threadIdx.x);
      
     
     /*Merge*/
      int K[2],P[2],Q[2];
      int offset;

      //if (i==0) printf("A: %d B:%d M:%d\n", size_A, size_B, size_M);
      if (i>size_A)
      {
          K[0]=i-size_A;
          K[1]=size_A;
          P[0]=size_A;
          P[1]=i-size_A;
      }
      else
      {
          K[0]=0;
          K[1]=i;
          P[0]=i;
          P[1]=0;
          //if (i==0) printf("K(%d,%d) et P(%d,%d)\n",K[0],K[1],P[0],P[1]);
      }
      while (1)
      {
          offset=abs(K[1]-P[1])/2;
          //if (i==0) printf("K(%d,%d) et P(%d,%d)\n",K[0],K[1],P[0],P[1]);
          Q[1]=K[1]-offset;
          Q[0]=K[0]+offset;
          if (Q[1] >= 0 && Q[0] <= size_B && (Q[1]== size_A || Q[0]==0 || A[Q[1]]>B[Q[0]-1]))//verif
          {
              if (Q[0]==size_B || Q[1]==0 || A[Q[1]-1]<=B[Q[0]])//verif
              {
                  if (Q[1]<size_A && (Q[0]==size_B || A[Q[1]]<=B[Q[0]]))//verif
                  {
                      M[blockIdx.x * slice_size + i]=A[Q[1]];
                   //printf("\n blockIdx = %d, threadIdx = %d : M[%d] = A[%d]\n",blockIdx.x,threadIdx.x,blockIdx.x * slice_size + i,Q[1]);
                  }
                  else
                  {
                      M[blockIdx.x * slice_size + i]=B[Q[0]]; 
                    //printf("\n blockIdx = %d, threadIdx = %d : M[%d] = B[%d]\n",blockIdx.x,threadIdx.x,blockIdx.x * slice_size + i,Q[0]);
                  }
                  break;
              }
              else
              {
                  K[0]=Q[0]+1;
                  K[1]=Q[1]-1;
              }
          }
          else
          {
            P[0]=Q[0]-1;
            P[1]=Q[1]+1;
          }
      }
    }
  //printf("pour %d on fait %d tours \n",blockIdx.x * blockDim.x + threadIdx.x,c);
}

__global__ void sortManager(int *M, int size_A, int size_B, int size_M, int number_of_slices)
{

        int slice_size = size_A + size_B;
        

       /*Chargement de A et B dans la shared memory*/
      /*Comme on a une seule shared memory*/
      __shared__ int shared_AB[1024];  //Comme A et B ne peuvent pas dépasser 1024
 
      int* s_A = (int*) &shared_AB[0];
      int* s_B = (int*) &s_A[size_A];
 
      __syncthreads();
 
      //int i = blockDim.x * blockIdx.x + threadIdx.x;
      if (threadIdx.x < size_A)
      {
          s_A[threadIdx.x] = M[blockIdx.x *slice_size+ threadIdx.x];
          //printf("\n blockIdx = %d, threadIdx = %d : s_A[%d] = M[%d] = %d\n",blockIdx.x,threadIdx.x,threadIdx.x, blockIdx.x *slice_size+ threadIdx.x, M[blockIdx.x *slice_size+ threadIdx.x]);
      }
        
 
      if (threadIdx.x >= size_A && threadIdx.x < size_B + size_A  )
      {
          s_B[threadIdx.x-size_A] = M[blockIdx.x *slice_size+ threadIdx.x]; 
          //printf("\n blockIdx = %d, threadIdx = %d : s_B[%d] = M[%d] = %d\n",blockIdx.x,threadIdx.x,threadIdx.x, blockIdx.x *slice_size+ threadIdx.x, M[blockIdx.x *slice_size+ threadIdx.x]);
      }
        
 

      

      __syncthreads();

      mergeSmallBatch_k(s_A, s_B, M, size_A, size_B, size_M,slice_size); 
    

    
}

__global__ void sortManager_extraSlice(int *M ,int size_A,int size_B,int size_A_extra,int size_B_extra,int size_M,int number_of_slices)
{

        int slice_size = size_A + size_B;
        
        if (threadIdx.x == 0)
        {
          printf(" small_batches true ");  
        }

          
  

       /*Chargement de A et B dans la shared memory*/
      /*Comme on a une seule shared memory*/
      __shared__ int shared_AB[1024];  //Comme A et B ne peuvent pas dépasser 1024
 
      int* s_A = (int*) &shared_AB[0];
     
      int* s_B;
     
      if (blockIdx.x == number_of_slices)
        s_B = (int*) &s_A[size_A_extra];

      else
        s_B = (int*) &s_A[size_A];
 
      __syncthreads();
 
      //int i = blockDim.x * blockIdx.x + threadIdx.x;
      if (threadIdx.x < size_A)
      {
          s_A[threadIdx.x] = M[blockIdx.x *slice_size+ threadIdx.x];
          //printf("\n blockIdx = %d, threadIdx = %d : s_A[%d] = M[%d] = %d\n",blockIdx.x,threadIdx.x,threadIdx.x, blockIdx.x *slice_size+ threadIdx.x, M[blockIdx.x *slice_size+ threadIdx.x]);
      }
        
 
      if (threadIdx.x >= size_A && threadIdx.x < size_B + size_A)
      {
          s_B[threadIdx.x-size_A] = M[blockIdx.x *slice_size+ threadIdx.x]; 
          //printf("\n blockIdx = %d, threadIdx = %d : s_B[%d] = M[%d] = %d\n",blockIdx.x,threadIdx.x,threadIdx.x, blockIdx.x *slice_size+ threadIdx.x, M[blockIdx.x *slice_size+ threadIdx.x]);
      }
     
      if (blockIdx.x == number_of_slices && threadIdx.x < size_A_extra)
      {
          s_A[threadIdx.x] = M[blockIdx.x *slice_size+ threadIdx.x];
      }
     
      if (blockIdx.x == number_of_slices && threadIdx.x >= size_A_extra && threadIdx.x < size_B_extra + size_A_extra)
      {
          s_B[threadIdx.x-size_A_extra] = M[blockIdx.x *slice_size+ threadIdx.x]; 
      }
     

      __syncthreads();

     if (blockIdx.x == number_of_slices)
     {
         if (size_A_extra < size_B_extra)
            mergeSmallBatch_k(s_B, s_A, M, size_B_extra, size_A_extra, size_M, slice_size);  
      
        else
          mergeSmallBatch_k(s_A, s_B, M, size_A_extra, size_B_extra, size_M, slice_size); 
   
     }

    else
      mergeSmallBatch_k(s_A, s_B, M, size_A, size_B, size_M,slice_size); 
    
 
}

 
//*****************************************************************************
//MAIN
//*****************************************************************************
int main(int argc, char *argv[]) {
    
  srand (time (NULL));
  /*Déclaration des variables CPU*/
  /*Taille des tableaux*/

  //int h_taille_M=rand()%(N-1)+1;  
  //int h_taille_M=14; 
  int h_taille_M=N; 

  printf("Taille de M : %d\n",h_taille_M);

  /*Tableaux et allocation memoire*/
  int *h_M;
  h_M=(int *)malloc(h_taille_M*sizeof(int));

  /*Déclaration des variables GPU*/ 
  int *d_M;
  hipMalloc(&d_M,h_taille_M*sizeof(int));

   /*Initialisation et preparation des tableaux*/
  for (int i=0; i<h_taille_M;i++)
    h_M[i]=rand()%10000;

/*

     printf("\n");
  printf("***M***\n");
  for (int i=0; i<h_taille_M; i=i+1)
    printf("%d\n",h_M[i]);

*/

    /*Transfert la mémoire du cpu vers le gpu*/
  hipMemcpy(d_M, h_M, h_taille_M*sizeof(int), hipMemcpyHostToDevice);

  /*Merge tableau*/
  int h_slice_size=1;
  int h_number_of_slices=h_taille_M/2;
  int h_slice_reste_precedent=0;
  int h_slice_reste=0;

  while (h_number_of_slices > 0)
  {   
      /*Mise a jour taille et indices*/
      h_slice_size=2*h_slice_size;
      
      h_slice_reste_precedent=h_slice_reste;
      h_slice_reste=h_taille_M%h_slice_size;
      h_number_of_slices=h_taille_M/h_slice_size;

      if (h_slice_reste_precedent!=0 && h_slice_reste!=0)
      {
          int h_taille_A_extra=h_slice_reste-h_slice_reste_precedent;
          int h_taille_B_extra=h_slice_reste_precedent;
          sortManager_extraSlice<<<h_number_of_slices+1,threadsPerBlock>>>(d_M,h_slice_size/2,h_slice_size/2,h_taille_A_extra,h_taille_B_extra,h_taille_M,h_number_of_slices);

      }

      sortManager<<<h_number_of_slices,threadsPerBlock>>>(d_M, h_slice_size/2, h_slice_size/2,h_slice_size,h_number_of_slices);



  }

  /*Transfert la mémoire du gpu vers le cpu*/
  hipMemcpy(h_M, d_M, h_taille_M*sizeof(int), hipMemcpyDeviceToHost);

  /*Affichage du resultat*/
  
  /*
  printf("\n");
  printf("***M***\n");
  for (int i=0; i<h_taille_M; i=i+1)
    printf("%d\n",h_M[i]);
  
  */


  if (verif_trie(h_M,h_taille_M)==1)
    printf("\n ok tableau trié\n");
    else
  {
    printf("\n KO probleme a l indice %d\n",verif_trie(h_M,h_taille_M));
    //printf("%d %d %d %d\n",h_M[verif_trie(h_M,h_taille_M)-1],h_M[verif_trie(h_M,h_taille_M)], h_M[verif_trie(h_M,h_taille_M)+1], h_M[verif_trie(h_M,h_taille_M)+2] );
  }

  hipFree(d_M);
  free(h_M);
 

    return 0;
}