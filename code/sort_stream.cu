
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>
#define N 100000
#define threadsPerBlock 1024
#define numBlock 65535


//*****************************************************************************
//Fonctions CPU de verification
//*****************************************************************************

int verif_trie(int *tab,int size)
{
    for (int i=0; i<size-1; i=i+1)
      if (tab[i]>tab[i+1])
          return i;
    return 1;
    
}


//*****************************************************************************
//Fonctions GPU (merge tableau)
//*****************************************************************************

__device__ void pathBig_k(int *A, int *B, int *Path, int size_A, int size_B, int size_M)
{
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i<size_M; i = i+blockDim.x*gridDim.x)
    {
      int K[2],P[2],Q[2];
      int offset;

      if (i>size_A)
      {
          K[0]=i-size_A;
          K[1]=size_A;
          P[0]=size_A;
          P[1]=i-size_A;
      }
      else
      {
          K[0]=0;
          K[1]=i;
          P[0]=i;
          P[1]=0;
         
      }
      while (1)
      {
          offset=abs(K[1]-P[1])/2;
          Q[1]=K[1]-offset;
          Q[0]=K[0]+offset;
          if (Q[1] >= 0 && Q[0] <= size_B && (Q[1]== size_A || Q[0]==0 || A[Q[1]]>B[Q[0]-1]))
          {
              if (Q[0]==size_B || Q[1]==0 || A[Q[1]-1]<=B[Q[0]])
              {
                  if (Q[1]<size_A && (Q[0]==size_B || A[Q[1]]<=B[Q[0]]))
                  {
                      Path[i]=1;
                      Path[i+size_M]=Q[1];
                  }
                  else
                  {
                      Path[i]=0;
                      Path[i+size_M]=Q[0];
                  }
                  break;
              }
              else
              {
                  K[0]=Q[0]+1;
                  K[1]=Q[1]-1;
              }
          }
          else
          {
            P[0]=Q[0]-1;
            P[1]=Q[1]+1;
          }
      }
    }
}


__device__ void mergeBig_k(int *A, int *B, int *M,int *Path, int size_A, int size_B, int size_M)
{
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i<size_M; i = i+blockDim.x*gridDim.x)
    {
        if (Path[i]==1)
          M[i]=A[Path[i+size_M]];
        else if (Path[i]==0)
          M[i]=B[Path[i+size_M]];
        else
          printf("ERROR thread num %d block %d",i,blockIdx.x);
                  
    }
}

__global__ void sortManager_GPU(int *A, int *B, int *M,int *Path, int size_A, int size_B, int size_M)
{
    pathBig_k(A, B, Path, size_A, size_B, size_M);
    mergeBig_k(A, B, M, Path, size_A, size_B, size_M);
}

void sortManager_CPU(int *h_M,int h_size_A,int h_size_B,int h_slice_size,int i,hipStream_t stream[])
{
    
    /*Variables CPU*/ 
    int h_size_M_tmp = h_size_A+h_size_B;
    int *h_A;
    int *h_B;
    int *h_M_tmp;
    h_A=(int *)malloc(h_size_A*sizeof(int));
    h_B=(int *)malloc(h_size_B*sizeof(int));
    h_M_tmp=(int *)malloc(h_size_M_tmp*sizeof(int));

    /*Remplir A et B*/
    for (int j=0; j<h_size_A; j++)
      h_A[j] = h_M[i*h_slice_size+j];
       
    for (int j=0; j<h_size_B; j++)
      h_B[j] = h_M[i*h_slice_size+j+h_size_A];

 
    /*Variables GPU*/
    int *d_A;
    int *d_B;
    int *d_M_tmp;
    int *d_Path_tmp;
    hipMalloc(&d_A,h_size_A*sizeof(int));
    hipMalloc(&d_B,h_size_B*sizeof(int));
    hipMalloc(&d_M_tmp,h_size_M_tmp*sizeof(int));
    hipMalloc(&d_Path_tmp,h_size_M_tmp*sizeof(int));

  
    /*Transfert*/
    hipMemcpyAsync(d_A, h_A, h_size_A*sizeof(int), hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync(d_B, h_B, h_size_B*sizeof(int), hipMemcpyHostToDevice, stream[i]);
 
    /*Sort d une slice de M*/
    if (h_size_A<h_size_B)
    {

        sortManager_GPU<<<numBlock,threadsPerBlock,0, stream[i]>>>(d_B, d_A, d_M_tmp, d_Path_tmp, h_size_B, h_size_A, h_size_M_tmp);
    }
    else
    {

        sortManager_GPU<<<numBlock,threadsPerBlock,0, stream[i]>>>(d_A, d_B, d_M_tmp, d_Path_tmp, h_size_A, h_size_B, h_size_M_tmp);   
    }
    
    /*Transfert memoire GPU*/
    hipMemcpyAsync(h_M_tmp, d_M_tmp, h_size_M_tmp*sizeof(int), hipMemcpyDeviceToHost, stream[i]);

    /*Copie de h_M_tmp dans h_M*/
    for (int j=0; j<h_size_M_tmp; j++)
      h_M[i*h_slice_size+j]=h_M_tmp[j];
 
    
    /*Liberation*/
    free(h_A);
    free(h_B);
    free(h_M_tmp);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_M_tmp);
    hipFree(d_Path_tmp);
}
//*****************************************************************************
//MAIN
//*****************************************************************************
int main() {

  srand (42);

  /*Declaration des variables CPU*/
  /*Taille des tableaux*/
  int h_taille_M=N; 

  /*Tableaux et allocation memoire*/
  int *h_M;
  h_M=(int *)malloc(h_taille_M*sizeof(int));
 
  /*Declaration des variables GPU*/  
  int *d_M;
  hipMalloc(&d_M,h_taille_M*sizeof(int));

  /*Initialisation et preparation des tableaux*/
  for (int i=0; i<h_taille_M;i++)
    h_M[i]=rand()%10000;

  /*Timer*/
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


  /*Sort de M*/
  int h_slice_size=1;
  int h_slice_number=h_taille_M/2;
  int h_slice_reste_precedent=0;
  int h_slice_reste=0;

  /*Declaration et creation des streams*/
  hipStream_t stream[h_slice_number+1];

  for (int ind_stream=0; ind_stream<h_slice_number+1; ind_stream++)
      hipStreamCreate(&stream[ind_stream]);  
      
  hipEventRecord(start);
  while (h_slice_number > 0)
  {   
      
      /*Mise a jour taille et indices*/
      h_slice_size=2*h_slice_size;
      
      /*Destruction des streams qui ne servent plus*/
      for (int i=(h_taille_M/h_slice_size)+1; i<h_slice_number+1; i++)
        hipStreamDestroy(stream[i]);
      
      /*Mise a jour taille et indices suite*/
      h_slice_reste_precedent=h_slice_reste;
      h_slice_reste=h_taille_M%h_slice_size;
      h_slice_number=h_taille_M/h_slice_size;
      
      
      for (int i=0; i<h_slice_number; i++) 
          sortManager_CPU(h_M,h_slice_size/2,h_slice_size/2,h_slice_size,i, stream);
          
      if (h_slice_reste_precedent!=0 && h_slice_reste!=0)
      {
              int h_taille_A=h_slice_reste-h_slice_reste_precedent;
              int h_taille_B=h_slice_reste_precedent;
              sortManager_CPU(h_M,h_taille_A,h_taille_B,h_slice_size,h_slice_number,stream);

      }
      hipDeviceSynchronize();

       
  }

  hipDeviceSynchronize();
  hipEventRecord(stop);

  /*Affichage du chrono*/
  hipEventSynchronize(stop);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  fprintf(stderr,"question3 stream Taille_M: %d, nbthreads: %d, numblocks: %d, Temps: %.5f, verif: %d\n", h_taille_M, threadsPerBlock, numBlock, ms,verif_trie(h_M,h_taille_M));
  

  /*Destruction des streams restants*/
  for (int i=0; i<h_slice_number; i++)
    hipStreamDestroy(stream[i]);

  /*Verification*/
  if (verif_trie(h_M,h_taille_M)==1)
    printf("ok tableau trie");
  else
    printf("KO recommencer %d ",verif_trie(h_M,h_taille_M) );
  
  /*Liberation*/
  free(h_M);


    return 0;
}