//*****************************************************************************
//Projet HPC fusion et trie de tableaux sur GPU
//Auteur: ROBIN Clement et SAULNIER Solene
//Promo: MAIN5
//Date: decembre 2020
//Question 5 en sequentiel
//*****************************************************************************


#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>
#define N 536870912
#define threadsPerBlock 1024
#define numBlocks 65535


//*****************************************************************************
//Fonctions CPU sort et verification
//*****************************************************************************
int verif_trie(int *tab,int size)
{
    for (int i=0; i<size-1; i=i+1)
      if (tab[i]>tab[i+1])
          return i;
    return -1;
    
}


void mergeSmall_k(int *A, int *B, int *M, int size_A, int size_B, int size_M)
{
  
  int i = 0;
  int j = 0;
  while (i+j<size_M)
  {  
    if (i>=size_A)
    {
      M[i+j]=B[j];
      j++;
    }
    else
    {
      if (j>=size_B || A[i]<B[j])
      {
        M[i+j]=A[i];
        i++;
      }
      else
      {
        M[i+j]=B[j];
        j++;
      }
    }
  }
}


void sortManager_CPU(int *h_M,int h_size_A,int h_size_B,int h_slice_size,int i)
{
    
    /*Variables CPU*/ 
    int h_size_M_tmp= h_size_A+h_size_B;
    int *h_A;
    int *h_B;
    int *h_M_tmp;
    h_A=(int *)malloc(h_size_A*sizeof(int));
    h_B=(int *)malloc(h_size_B*sizeof(int));
    h_M_tmp=(int *)malloc(h_size_M_tmp*sizeof(int));

    /*Remplir A et B*/
    for (int j=0; j<h_size_A; j++)
      h_A[j] = h_M[i*h_slice_size+j];

    for (int j=0; j<h_size_B; j++)
      h_B[j] = h_M[i*h_slice_size+j+h_size_A];
 
    /*Sort*/
    if (h_size_A<h_size_B)
      mergeSmall_k(h_B, h_A, h_M_tmp, h_size_B, h_size_A, h_size_M_tmp);
    else
      mergeSmall_k(h_A, h_B, h_M_tmp, h_size_A, h_size_B, h_size_M_tmp);   
    
    
    /*Copie de h_M_tmp dans h_M*/
    for (int j=0; j<h_size_M_tmp; j++)
      h_M[i*h_slice_size+j]=h_M_tmp[j];
 
    
    /*Liberation*/
    free(h_A);
    free(h_B);
    free(h_M_tmp);

}


//*****************************************************************************
//MAIN
//*****************************************************************************
int main(int argc, char const *argv[])
{
  //srand (time (NULL));
  srand (42);


  /*Déclaration des variables CPU*/
  /*Taille des tableaux*/
  int h_taille_M=1024*8;

  /*Traitement des options*/
  for (int i=0; i<argc-1; i=i+1)
  {
      if (strcmp(argv[i],"--s")==0 && atoi(argv[i+1])<N )
          h_taille_M=atoi(argv[i+1]);   
  }

  /*Tableaux et allocation memoire*/
  int *h_M;
  h_M=(int *)malloc(h_taille_M*sizeof(int));
 

  /*Initialisation et preparation des tableaux*/
  for (int i=0; i<h_taille_M;i++)
    h_M[i]=rand()%10000;

  /*Merge tableau*/
  /*variables generales*/
  int h_slice_size=1;
  int h_number_of_slices=h_taille_M/h_slice_size;
  int h_slice_reste_precedent=0;
  int h_slice_reste=0;
  
  /*Timer*/
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /*Mise a jour taille et indices suite*/    
  hipEventRecord(start);
  while (h_number_of_slices>0)
  {   
      /*Mise a jour taille et indices*/
      h_slice_size=2*h_slice_size;
      /*Mise a jour taille et indices suite*/
      h_slice_reste_precedent=h_slice_reste;
      h_slice_reste=h_taille_M%h_slice_size;
      h_number_of_slices=h_taille_M/h_slice_size;
      
      
      for (int i=0; i<h_number_of_slices; i++)
      {   
          sortManager_CPU(h_M,h_slice_size/2,h_slice_size/2,h_slice_size,i);
          
      }
      if (h_slice_reste_precedent!=0 && h_slice_reste!=0)
      {
          int h_taille_A=h_slice_reste-h_slice_reste_precedent;
          int h_taille_B=h_slice_reste_precedent;
          sortManager_CPU(h_M,h_taille_A,h_taille_B,h_slice_size,h_number_of_slices);

      }
       
  }
  
  hipEventRecord(stop);

  /*Affichage du chrono*/
  hipEventSynchronize(stop);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  fprintf(stderr,"mergeBatches_seq Taille_M: %d, nbthreads: %d, numblocks: %d, Temps: %.5f, verif: %d\n", h_taille_M, threadsPerBlock, numBlocks, ms,verif_trie(h_M,h_taille_M));
  
  /*Verification*/
  if (verif_trie(h_M,h_taille_M)==-1)
    printf("ok tableau trie");
  else
    printf("KO recommencer %d ",verif_trie(h_M,h_taille_M) );

  /*Liberation*/    
  free(h_M);


    return 0;
}