
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>
#include <string.h>
#define N 1024

//*****************************************************************************
//Fonctions CPU fusion et verification
//*****************************************************************************
int verif_trie(int *tab,int size)
{
    for (int i=0; i<size-1; i=i+1)
      if (tab[i]>tab[i+1])
          return i;
    return 1;
    
}

void fusion(int* tableau,int deb1,int fin1,int fin2)
{
  int *table1;
  int deb2=fin1+1;
  int compt1=deb1;
  int compt2=deb2;
  int i;
        
  table1=(int *) malloc((fin1-deb1+1)*sizeof(int));

  //on recopie les éléments du début du tableau
  for(i=deb1;i<=fin1;i++)
  {
      table1[i-deb1]=tableau[i];
  }
                  
  for(i=deb1;i<=fin2;i++)
  {        
    if (compt1==deb2) //c'est que tous les éléments du premier tableau ont été utilisés
    {
      break; //tous les éléments ont donc été classés
    }
    else if (compt2==(fin2+1)) //c'est que tous les éléments du second tableau ont été utilisés
    {
      tableau[i]=table1[compt1-deb1]; //on ajoute les éléments restants du premier tableau
      compt1++;
    }
    else if (table1[compt1-deb1]<tableau[compt2])
    {
      tableau[i]=table1[compt1-deb1]; //on ajoute un élément du premier tableau
      compt1++;
    }
    else
    {
      tableau[i]=tableau[compt2]; //on ajoute un élément du second tableau
      compt2++;
    }
  }
  free(table1);
}
        

void tri_fusion_bis(int* tableau,int deb,int fin)
{
  if (deb!=fin)
  {
    int milieu=(fin+deb)/2;
    tri_fusion_bis(tableau,deb,milieu);
    tri_fusion_bis(tableau,milieu+1,fin);
    fusion(tableau,deb,milieu,fin);
  }
}

void tri_fusion(int* tableau,int longueur)
{
  if (longueur>0)
  {
    tri_fusion_bis(tableau,0,longueur-1);
  }
}

//*****************************************************************************
//Fonctions CPU (merge tableau)
//*****************************************************************************

void mergeSmall_k(int *A, int *B, int *M, int size_A, int size_B, int size_M)
{
  
  int i = 0;
  int j = 0;
  while (i+j<size_M)
  {  
    if (i>size_A)
    {
      M[i+j]=B[j];
      j++;
    }
    else
    {
      if (j>=size_B || A[i]<B[j])
      {
        M[i+j]=A[i];
        i++;
      }
      else
      {
        M[i+j]=B[j];
        j++;
      }
    }
  }
}


 
//*****************************************************************************
//MAIN
//*****************************************************************************
int main(int argc, char *argv[]) {

  srand(42);

  /*Déclaration des variables CPU*/
  /*Taille des tableaux*/
  int h_taille_A=N-10;
  int h_taille_B=N-h_taille_A;
  int h_taille_M=h_taille_A+h_taille_B;

  if (h_taille_A < h_taille_B)
  {
    int tpm=h_taille_A;
    h_taille_A=h_taille_B;
    h_taille_B=tpm;
  }

  /*Tableaux et allocation memoire*/
  int *h_A;
  int *h_B;
  int *h_M;
  h_A=(int *)malloc(h_taille_A*sizeof(int));
  h_B=(int *)malloc(h_taille_B*sizeof(int));
  h_M=(int *)malloc(h_taille_M*sizeof(int));

  
   
  /*Initialisation et preparation des tableaux*/
  for (int i=0; i<h_taille_A;i++)
    h_A[i]=rand()%10000;
  
  for (int i=0; i<h_taille_B;i++)
    h_B[i]=rand()%10000;
  
  tri_fusion(h_A, h_taille_A);
  tri_fusion(h_B, h_taille_B);

  /*Timer*/
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /*Merge tableau*/
  hipEventRecord(start);
  mergeSmall_k(h_A, h_B, h_M, h_taille_A, h_taille_B, h_taille_M);
  hipDeviceSynchronize();
  hipEventRecord(stop);

  /*Affichage du chrono*/
  hipEventSynchronize(stop);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  printf("\nCode sequentiel\n Taille_M: %d, Temps:%.5f\n", h_taille_M,ms);
  
  
  /*Verification*/
  if (verif_trie(h_M,h_taille_M)==1)
    printf("\n ok tableau M trié\n");
  else
    printf("\n KO probleme a l indice %d\n",verif_trie(h_M,h_taille_M));



  /*Liberation*/
  free(h_A);
  free(h_B);
  free(h_M);
 

    return 0;
} 