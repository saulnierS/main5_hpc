//*****************************************************************************
//Projet HPC fusion et trie de tableaux sur GPU
//Auteur: ROBIN Clement et SAULNIER Solene
//Promo: MAIN5
//Date: decembre 2020
//information code en sur cpu
//*****************************************************************************

#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>
#include <string.h>
#define N 536870912

//*****************************************************************************
//Fonctions CPU fusion et verification
//*****************************************************************************
int verif_trie(int *tab,int size)
{
    for (int i=0; i<size-1; i=i+1)
      if (tab[i]>tab[i+1])
          return i;
    return -1;
    
}

void fusion(int* tableau,int deb1,int fin1,int fin2)
{
  int *table1;
  int deb2=fin1+1;
  int compt1=deb1;
  int compt2=deb2;
  int i;
        
  table1=(int *) malloc((fin1-deb1+1)*sizeof(int));

  //on recopie les elements du debut du tableau
  for(i=deb1;i<=fin1;i++)
  {
      table1[i-deb1]=tableau[i];
  }
                  
  for(i=deb1;i<=fin2;i++)
  {        
    if (compt1==deb2) //c est que tous les elements du premier tableau ont ete utilises
    {
      break; //tous les elements ont donc ete classes
    }
    else if (compt2==(fin2+1)) //c est que tous les elements du second tableau ont ete utilises
    {
      tableau[i]=table1[compt1-deb1]; //on ajoute les elements restants du premier tableau
      compt1++;
    }
    else if (table1[compt1-deb1]<tableau[compt2])
    {
      tableau[i]=table1[compt1-deb1]; //on ajoute un element du premier tableau
      compt1++;
    }
    else
    {
      tableau[i]=tableau[compt2]; //on ajoute un element du second tableau
      compt2++;
    }
  }
  free(table1);
}
        

void tri_fusion_bis(int* tableau,int deb,int fin)
{
  if (deb!=fin)
  {
    int milieu=(fin+deb)/2;
    tri_fusion_bis(tableau,deb,milieu);
    tri_fusion_bis(tableau,milieu+1,fin);
    fusion(tableau,deb,milieu,fin);
  }
}

void tri_fusion(int* tableau,int longueur)
{
  if (longueur>0)
  {
    tri_fusion_bis(tableau,0,longueur-1);
  }
}

//*****************************************************************************
//Fonctions CPU (merge tableau)
//*****************************************************************************

void mergeSmall_k(int *A, int *B, int *M, int size_A, int size_B, int size_M)
{
  
  int i = 0;
  int j = 0;
  while (i+j<size_M)
  {  
    if (i>size_A)
    {
      M[i+j]=B[j];
      j++;
    }
    else
    {
      if (j>=size_B || A[i]<B[j])
      {
        M[i+j]=A[i];
        i++;
      }
      else
      {
        M[i+j]=B[j];
        j++;
      }
    }
  }
}


 
//*****************************************************************************
//MAIN
//*****************************************************************************
int main(int argc, char *argv[]) 
{
  //srand (time (NULL));
  srand(42);

  /*Declaration des variables CPU*/
  /*Taille des tableaux*/
  int h_taille_A=1024/2;
  int h_taille_B=1024-h_taille_A;
  int h_taille_M=h_taille_A+h_taille_B;

  /*Traitement des options*/
  for (int i=0; i<argc-1; i=i+1)
  {
      if (strcmp(argv[i],"--sA")==0 && atoi(argv[i+1])<N )
          h_taille_A=atoi(argv[i+1]);
      if (strcmp(argv[i],"--sB")==0 && atoi(argv[i+1])<N)
          h_taille_B=atoi(argv[i+1]);   
  }


  if (h_taille_A < h_taille_B)
  {
    int tpm=h_taille_A;
    h_taille_A=h_taille_B;
    h_taille_B=tpm;
  }

  /*Tableaux et allocation memoire*/
  int *h_A;
  int *h_B;
  int *h_M;
  h_A=(int *)malloc(h_taille_A*sizeof(int));
  h_B=(int *)malloc(h_taille_B*sizeof(int));
  h_M=(int *)malloc(h_taille_M*sizeof(int));
  
   
  /*Initialisation et preparation des tableaux*/
  for (int i=0; i<h_taille_A;i++)
    h_A[i]=rand()%10000;
  
  for (int i=0; i<h_taille_B;i++)
    h_B[i]=rand()%10000;
  
  tri_fusion(h_A, h_taille_A);
  tri_fusion(h_B, h_taille_B);

  /*Timer*/
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /*Merge tableau*/
  hipEventRecord(start);
  mergeSmall_k(h_A, h_B, h_M, h_taille_A, h_taille_B, h_taille_M);
  hipDeviceSynchronize();
  hipEventRecord(stop);

  /*Affichage du chrono*/
  hipEventSynchronize(stop);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  fprintf(stderr,"code sequentiel Taille_A: %d, Taille_B: %d, Taille_M: %d, Temps: %.5f, verif: %d\n", h_taille_A, h_taille_B, h_taille_M,ms,verif_trie(h_M,h_taille_M));
  
  
  /*Verification*/
  if (verif_trie(h_M,h_taille_M)==-1)
    printf("\n ok tableau M trie\n");
  else
    printf("\n KO probleme a l indice %d\n",verif_trie(h_M,h_taille_M));


  /*Liberation*/
  free(h_A);
  free(h_B);
  free(h_M);
 

    return 0;
} 