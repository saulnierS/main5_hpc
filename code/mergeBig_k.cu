//*****************************************************************************
//Projet HPC fusion et trie de tableaux sur GPU
//Auteur: ROBIN Clement et SAULNIER Solene
//Promo: MAIN5
//Date: decembre 2020
//Question 2
//*****************************************************************************



#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>
#define N 67107840
#define threadsPerBlock 1024
#define numBlock 65535


//*****************************************************************************
//Fonctions CPU fusion et verification
//*****************************************************************************
int verif_trie(int *tab,int size)
{
    for (int i=0; i<size-1; i=i+1)
      if (tab[i]>tab[i+1])
          return i;
    return -1;
    
}

void fusion(int* tableau,int deb1,int fin1,int fin2)
{
  int *table1;
  int deb2=fin1+1;
  int compt1=deb1;
  int compt2=deb2;
  int i;
        
  table1=(int *) malloc((fin1-deb1+1)*sizeof(int));

  //on recopie les elements du debut du tableau
  for(i=deb1;i<=fin1;i++)
  {
      table1[i-deb1]=tableau[i];
  }
                  
  for(i=deb1;i<=fin2;i++)
  {        
    if (compt1==deb2) //c'est que tous les elements du premier tableau ont ete utilises
    {
      break; //tous les elements ont donc ete classes
    }
    else if (compt2==(fin2+1)) //c'est que tous les elements du second tableau ont ete utilises
    {
      tableau[i]=table1[compt1-deb1]; //on ajoute les elements restants du premier tableau
      compt1++;
    }
    else if (table1[compt1-deb1]<tableau[compt2])
    {
      tableau[i]=table1[compt1-deb1]; //on ajoute un element du premier tableau
      compt1++;
    }
    else
    {
      tableau[i]=tableau[compt2]; //on ajoute un element du second tableau
      compt2++;
    }
  }
  free(table1);
}
        

void tri_fusion_bis(int* tableau,int deb,int fin)
{
  if (deb!=fin)
  {
    int milieu=(fin+deb)/2;
    tri_fusion_bis(tableau,deb,milieu);
    tri_fusion_bis(tableau,milieu+1,fin);
    fusion(tableau,deb,milieu,fin);
  }
}

void tri_fusion(int* tableau,int longueur)
{
  if (longueur>0)
  {
    tri_fusion_bis(tableau,0,longueur-1);
  }
}


//*****************************************************************************
//Fonctions GPU (merge tableau)
//*****************************************************************************

__device__ void pathBig_k(int *A, int *B, int *Path, int size_A, int size_B, int size_M)
{
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i<size_M; i = i+blockDim.x*gridDim.x)
    {
      int K[2],P[2],Q[2];
      int offset;

      if (i>size_A)
      {
          K[0]=i-size_A;
          K[1]=size_A;
          P[0]=size_A;
          P[1]=i-size_A;
      }
      else
      {
          K[0]=0;
          K[1]=i;
          P[0]=i;
          P[1]=0;
      }
      while (1)
      {
          offset=abs(K[1]-P[1])/2;
          Q[1]=K[1]-offset;
          Q[0]=K[0]+offset;
          if (Q[1] >= 0 && Q[0] <= size_B && (Q[1]== size_A || Q[0]==0 || A[Q[1]]>B[Q[0]-1]))
          {
              if (Q[0]==size_B || Q[1]==0 || A[Q[1]-1]<=B[Q[0]])
              {
                  if (Q[1]<size_A && (Q[0]==size_B || A[Q[1]]<=B[Q[0]]))
                  {
                      Path[i]=1;
                      Path[i+size_M]=Q[1];
                  }
                  else
                  {
                      Path[i]=0;
                      Path[i+size_M]=Q[0];
                  }
                  break;
              }
              else
              {
                  K[0]=Q[0]+1;
                  K[1]=Q[1]-1;
              }
          }
          else
          {
            P[0]=Q[0]-1;
            P[1]=Q[1]+1;
          }
      }
    }
}


__device__ void mergeBig_k(int *A, int *B, int *M,int *Path, int size_A, int size_B, int size_M)
{
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i<size_M; i = i+blockDim.x*gridDim.x)
    {
        if (Path[i]==1)
          M[i]=A[Path[i+size_M]];
        else if (Path[i]==0)
          M[i]=B[Path[i+size_M]];
        else
          printf("ERROR thread num %d block %d",i,blockIdx.x);
                  
    }
}

__global__ void sortManager(int *A, int *B, int *M, int *Path, int size_A, int size_B, int size_M)
{

    pathBig_k(A, B, Path, size_A, size_B, size_M);
    mergeBig_k(A, B, M, Path, size_A, size_B, size_M);
}
 
//*****************************************************************************
//MAIN
//*****************************************************************************
int main(int argc, char const *argv[])
{

  srand (42);
  int numThreads=threadsPerBlock;
  /*Declaration des variables CPU*/
  /*Taille des tableaux*/
  int h_taille_A=rand()%(N-1)+1;
  int h_taille_B=N-h_taille_A;
  int h_taille_M=h_taille_A+h_taille_B;

  if (h_taille_A < h_taille_B)
  {
      int tpm=h_taille_A;
      h_taille_A=h_taille_B;
      h_taille_B=tpm;
  }


  /*Tableaux et allocation memoire*/
  int *h_A;
  int *h_B;
  int *h_M;
  int *h_Path;
  h_A=(int *)malloc(h_taille_A*sizeof(int));
  h_B=(int *)malloc(h_taille_B*sizeof(int));
  h_M=(int *)malloc(h_taille_M*sizeof(int));
  h_Path=(int *)malloc(2*h_taille_M*sizeof(int));
 

  /*Declaration des variables GPU*/ 
  int *d_A; 
  int *d_B; 
  int *d_M;
  int *d_Path;
  hipMalloc(&d_A,h_taille_A*sizeof(int)); 
  hipMalloc(&d_B,h_taille_B*sizeof(int));
  hipMalloc(&d_M,h_taille_M*sizeof(int));
  hipMalloc(&d_Path,2*h_taille_M*sizeof(int));
    

   
  /*Initialisation et preparation des tableaux*/
  for (int i=0; i<h_taille_A;i++)
    h_A[i]=rand()%10000;
  for (int i=0; i<h_taille_B;i++)
    h_B[i]=rand()%10000;
  
  
  tri_fusion(h_A, h_taille_A);
  tri_fusion(h_B, h_taille_B);


  /*Transfert la memoire du cpu vers le gpu*/
  hipMemcpy(d_A, h_A, h_taille_A*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, h_taille_B*sizeof(int), hipMemcpyHostToDevice);


  /*Timer*/
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


  /*Merge tableau*/
  hipEventRecord(start);
  sortManager<<<numBlock,threadsPerBlock>>>(d_A, d_B, d_M, d_Path, h_taille_A, h_taille_B, h_taille_M);
  hipDeviceSynchronize();
  hipEventRecord(stop);


  /*Transfert la memoire du gpu vers le cpu*/
  hipMemcpy(h_M, d_M, h_taille_M*sizeof(int), hipMemcpyDeviceToHost);


  /*Affichage du chrono*/
  hipEventSynchronize(stop);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  fprintf(stderr,"mergeBig_k Taille_A: %d, Taille_B: %d, Taille_M: %d, nbthreads: %d, numblocks: %d, Temps: %.5f, verif: %d\n", h_taille_A, h_taille_B, h_taille_M,numThreads,numBlock,ms,verif_trie(h_M,h_taille_M));


  if (verif_trie(h_M,h_taille_M)==-1)
    printf("\nok tableau trie\n");
  else
    printf("\nKO probleme a l indice %d\n",verif_trie(h_M,h_taille_M));

  /*Liberation*/
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_M);
  hipFree(d_Path);
  free(h_A);
  free(h_B);
  free(h_M);
  free(h_Path);
 

    return 0;
} 