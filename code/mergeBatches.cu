//*****************************************************************************
//Projet HPC fusion et trie de tableaux sur GPU
//Auteur: ROBIN Clement et SAULNIER Solene
//Promo: MAIN5
//Date: decembre 2020
//Question 5 avec streams et utilisation de la shared
//*****************************************************************************


#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>
#define N 536870912 //taille max du tableau =d dans le projet
#define threadsPerBlock 1024
#define numBlocks 65535


//*****************************************************************************
//Fonctions CPU fusion et verification
//*****************************************************************************
int verif_trie(int *tab,int size)
{
    for (int i=0; i<size-1; i=i+1)
      if (tab[i]>tab[i+1])
          return i;
    return -1;
    
}

//*****************************************************************************
//Fonctions GPU (merge tableau) small
//*****************************************************************************
__device__ void mergeSmallBatch_k(int *A, int *B, int *M, int size_A, int size_B, int size_M, int slice_size)
{

    int i = threadIdx.x;
    if (i < size_A+size_B)
    {

     /*Merge*/
      int K[2],P[2],Q[2];
      int offset;

      if (i>size_A)
      {
          K[0]=i-size_A;
          K[1]=size_A;
          P[0]=size_A;
          P[1]=i-size_A;
      }
      else
      {
          K[0]=0;
          K[1]=i;
          P[0]=i;
          P[1]=0;
      }
      while (1)
      {
          offset=abs(K[1]-P[1])/2;
          Q[1]=K[1]-offset;
          Q[0]=K[0]+offset;
          if (Q[1] >= 0 && Q[0] <= size_B && (Q[1]== size_A || Q[0]==0 || A[Q[1]]>B[Q[0]-1]))
          {
              if (Q[0]==size_B || Q[1]==0 || A[Q[1]-1]<=B[Q[0]])
              {
                  if (Q[1]<size_A && (Q[0]==size_B || A[Q[1]]<=B[Q[0]]))
                      M[blockIdx.x * slice_size + i]=A[Q[1]];
                   
                  else
                      M[blockIdx.x * slice_size + i]=B[Q[0]]; 
                   
                  break;
              }
              else
              {
                  K[0]=Q[0]+1;
                  K[1]=Q[1]-1;
              }
          }
          else
          {
            P[0]=Q[0]-1;
            P[1]=Q[1]+1;
          }
      }
    }

}

__global__ void small_sortManager(int *M, int size_A, int size_B, int size_M, int number_of_slices)
{

      int slice_size = size_A + size_B;
        

      /*Chargement de A et B dans la shared memory*/
      /*Comme on a une seule shared memory*/
      __shared__ int shared_AB[1024];  //Comme A et B ne peuvent pas dépasser 1024
 
      int* s_A = (int*) &shared_AB[0];
      int* s_B = (int*) &s_A[size_A];
 
      __syncthreads();
 
      if (threadIdx.x < size_A)
        s_A[threadIdx.x] = M[blockIdx.x *slice_size+ threadIdx.x];

        

      if (threadIdx.x >= size_A && threadIdx.x < size_B + size_A  )
        s_B[threadIdx.x-size_A] = M[blockIdx.x *slice_size+ threadIdx.x]; 


      __syncthreads();
      /*if (size_A==1 && size_B==1)
      {
          //swap pour 2 elements
          if (threadIdx.x==0)
          {
              if (s_A[0]>s_B[0])
              {
                  M[blockIdx.x * 2]=s_B[0];
                  M[blockIdx.x * 2+1]=s_A[0];
              }
              else
              {
                  M[blockIdx.x * 2]=s_A[0];
                  M[blockIdx.x * 2+1]=s_B[0];
              }
          }
      }
      else*/ 
        mergeSmallBatch_k(s_A, s_B, M, size_A, size_B, size_M,slice_size);    
}

__global__ void small_sortManager_extraSlice(int *M ,int size_A,int size_B,int size_A_extra,int size_B_extra,int size_M,int number_of_slices)
{

        int slice_size = size_A + size_B;

      /*Chargement de A et B dans la shared memory*/
      /*Comme on a une seule shared memory*/
      __shared__ int shared_AB[1024];  //Comme A et B ne peuvent pas dépasser 1024
 
      int* s_A = (int*) &shared_AB[0];
     
      int* s_B;
     
      if (blockIdx.x == number_of_slices)
        s_B = (int*) &s_A[size_A_extra];

      else
        s_B = (int*) &s_A[size_A];
 
      __syncthreads();
 
      if (threadIdx.x < size_A)
        s_A[threadIdx.x] = M[blockIdx.x *slice_size+ threadIdx.x];

 
      if (threadIdx.x >= size_A && threadIdx.x < size_B + size_A)
        s_B[threadIdx.x-size_A] = M[blockIdx.x *slice_size+ threadIdx.x]; 

     
      if (blockIdx.x == number_of_slices && threadIdx.x < size_A_extra)
        s_A[threadIdx.x] = M[blockIdx.x *slice_size+ threadIdx.x];
     
      if (blockIdx.x == number_of_slices && threadIdx.x >= size_A_extra && threadIdx.x < size_B_extra + size_A_extra)
        s_B[threadIdx.x-size_A_extra] = M[blockIdx.x *slice_size+ threadIdx.x]; 
     

      __syncthreads();

     if (blockIdx.x == number_of_slices)
     {
         /*
         if (size_A_extra==1 && size_B_extra==1)
         {
          //swap pour 2 elements
          if (threadIdx.x==0)
          {
              if (s_A[0]>s_B[0])
              {
                  M[blockIdx.x * 2]=s_B[0];
                  M[blockIdx.x * 2+1]=s_A[0];
              }
              else
              {
                  M[blockIdx.x * 2]=s_A[0];
                  M[blockIdx.x * 2+1]=s_B[0];
              }
            }
          }
          else*/
          //{ 
              if (size_A_extra < size_B_extra)
                mergeSmallBatch_k(s_B, s_A, M, size_B_extra, size_A_extra, size_M, slice_size);  
      
              else
                mergeSmallBatch_k(s_A, s_B, M, size_A_extra, size_B_extra, size_M, slice_size); 
          //}
     }

    else
    {
      /*if (size_A==1 && size_B==1)
      {
          //swap pour 2 elements
          if (threadIdx.x==0)
          {
              if (s_A[0]>s_B[0])
              {
                  M[blockIdx.x * 2]=s_B[0];
                  M[blockIdx.x * 2+1]=s_A[0];
              }
              else
              {
                  M[blockIdx.x * 2]=s_A[0];
                  M[blockIdx.x * 2+1]=s_B[0];
              }
          }
      }
      else */
         mergeSmallBatch_k(s_A, s_B, M, size_A, size_B, size_M,slice_size); 
    }
      
    
 
}

//*****************************************************************************
//Fonctions GPU (merge tableau) big
//*****************************************************************************

__device__ void pathBig_k(int *A, int *B, int *Path, int size_A, int size_B, int size_M)
{
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i<size_M; i = i+blockDim.x*gridDim.x)
    {
      int K[2],P[2],Q[2];
      int offset;

      if (i>size_A)
      {
          K[0]=i-size_A;
          K[1]=size_A;
          P[0]=size_A;
          P[1]=i-size_A;
      }
      else
      {
          K[0]=0;
          K[1]=i;
          P[0]=i;
          P[1]=0;
      }
      while (1)
      {
          offset=abs(K[1]-P[1])/2;
          Q[1]=K[1]-offset;
          Q[0]=K[0]+offset;
          if (Q[1] >= 0 && Q[0] <= size_B && (Q[1]== size_A || Q[0]==0 || A[Q[1]]>B[Q[0]-1]))
          {
              if (Q[0]==size_B || Q[1]==0 || A[Q[1]-1]<=B[Q[0]])
              {
                  if (Q[1]<size_A && (Q[0]==size_B || A[Q[1]]<=B[Q[0]]))
                  {
                      Path[i]=1;
                      Path[i+size_M]=Q[1];
                  }
                  else
                  {
                      Path[i]=0;
                      Path[i+size_M]=Q[0];
                  }
                  break;
              }
              else
              {
                  K[0]=Q[0]+1;
                  K[1]=Q[1]-1;
              }
          }
          else
          {
            P[0]=Q[0]-1;
            P[1]=Q[1]+1;
          }
      }
    }
}


__device__ void mergeBig_k(int *A, int *B, int *M,int *Path, int size_A, int size_B, int size_M)
{
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i<size_M; i = i+blockDim.x*gridDim.x)
    {
        if (Path[i]==1)
          M[i]=A[Path[i+size_M]];
        else if (Path[i]==0)
          M[i]=B[Path[i+size_M]];
        else
          printf("ERROR thread num %d block %d",i,blockIdx.x);
                  
    }
}

__global__ void sortManager_GPU(int *A, int *B, int *M,int *Path, int size_A, int size_B, int size_M)
{
    pathBig_k(A, B, Path, size_A, size_B, size_M);
    mergeBig_k(A, B, M, Path, size_A, size_B, size_M);
}
//****************************************************************************************************
// Fonctions CPU
//****************************************************************************************************
void sortManager_CPU(int *h_M,int h_size_A,int h_size_B,int h_slice_size,int i,hipStream_t stream[])
{
    
    /*Variables CPU*/ 
    int h_size_M_tmp= h_size_A+h_size_B;
    int *h_A;
    int *h_B;
    int *h_M_tmp;
    h_A=(int *)malloc(h_size_A*sizeof(int));
    h_B=(int *)malloc(h_size_B*sizeof(int));
    h_M_tmp=(int *)malloc(h_size_M_tmp*sizeof(int));

    /*Remplir A et B*/
    for (int j=0; j<h_size_A; j++)
      h_A[j] = h_M[i*h_slice_size+j];

    for (int j=0; j<h_size_B; j++)
      h_B[j] = h_M[i*h_slice_size+j+h_size_A];
 
    /*Variables GPU*/
    int *d_A;
    int *d_B;
    int *d_M_tmp;
    int *d_Path_tmp;
    hipMalloc(&d_A,h_size_A*sizeof(int));
    hipMalloc(&d_B,h_size_B*sizeof(int));
    hipMalloc(&d_M_tmp,h_size_M_tmp*sizeof(int));
    hipMalloc(&d_Path_tmp,h_size_M_tmp*sizeof(int));

  
    /*Transfert*/
    hipMemcpyAsync(d_A, h_A, h_size_A*sizeof(int), hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync(d_B, h_B, h_size_B*sizeof(int), hipMemcpyHostToDevice, stream[i]);
 
    /*Sort*/
    if (h_size_A<h_size_B)
      sortManager_GPU<<<numBlocks,threadsPerBlock,0, stream[i]>>>(d_B, d_A, d_M_tmp, d_Path_tmp, h_size_B, h_size_A, h_size_M_tmp);
    else
      sortManager_GPU<<<numBlocks,threadsPerBlock,0, stream[i]>>>(d_A, d_B, d_M_tmp, d_Path_tmp, h_size_A, h_size_B, h_size_M_tmp);   
    
    /*Transfert memoire GPU*/
    hipMemcpyAsync(h_M_tmp, d_M_tmp, h_size_M_tmp*sizeof(int), hipMemcpyDeviceToHost, stream[i]);

    
    /*Copie de h_M_tmp dans h_M*/
    for (int j=0; j<h_size_M_tmp; j++)
      h_M[i*h_slice_size+j]=h_M_tmp[j];
 
    
    /*Liberation*/
    free(h_A);
    free(h_B);
    free(h_M_tmp);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_M_tmp);
    hipFree(d_Path_tmp);
}


//*****************************************************************************
//MAIN
//*****************************************************************************
int main(int argc, char const *argv[])
{
  //srand (time (NULL));
  srand (42);


  /*Déclaration des variables CPU*/
  /*Taille des tableaux*/
  int h_taille_M=N;

  /*Traitement des options*/
  for (int i=0; i<argc-1; i=i+1)
  {
      if (strcmp(argv[i],"--s")==0 && atoi(argv[i+1])<N )
          h_taille_M=atoi(argv[i+1]);   
  }

  /*Tableaux et allocation memoire*/
  int *h_M;
  h_M=(int *)malloc(h_taille_M*sizeof(int));
 
  /*Déclaration des variables GPU*/  
  int *d_M;
  hipMalloc(&d_M,h_taille_M*sizeof(int));

  /*Initialisation et preparation des tableaux*/
  for (int i=0; i<h_taille_M;i++)
    h_M[i]=rand()%10000;

  /*Merge tableau*/
  /*variables generales*/
  int h_slice_size=1;
  int h_number_of_slices=1024/2;
  int h_slice_reste_precedent=0;
  int h_slice_reste=0;
  
  /*Cas tailles de moins de 1024*/
  /*variables pour moins de 1024*/
  int h_is_irregular_batch=0;
  int h_irregular_batch_size=0;
  int h_irregular_slice_size=1;
  int h_irregular_number_of_slices=h_irregular_batch_size/2;
  int h_irregular_slice_reste_precedent=0;
  int h_irregular_slice_reste=0;


  /*Decoupage de M en batches de 1024*/
  int h_number_of_batches=h_taille_M/1024;
  if (h_taille_M%1024!=0)
  {
      h_irregular_batch_size=h_taille_M%1024;
      h_is_irregular_batch=1;
      h_irregular_number_of_slices=h_irregular_batch_size/2;   
  }
      

  /*Allocation et initialisation des batches*/
  /*Batches CPU*/
  int **h_batch_M;
  int *h_irregular_batch_M;
  h_batch_M = (int **) malloc( h_number_of_batches* sizeof(int *) );


  for (int b=0; b<h_number_of_batches; b++)
  {
    h_batch_M[b]=(int *) malloc (1024 * sizeof(int ));
    for (int ind=0; ind<1024; ind++)
        h_batch_M[b][ind]=h_M[b*1024+ind]; 
  }
  h_irregular_batch_M = (int *) malloc( h_irregular_batch_size* sizeof(int ) );
  for (int ind=0; ind<h_irregular_batch_size; ind++)
        h_irregular_batch_M[ind]=h_M[h_number_of_batches*1024+ind]; 
  
  /*Batches GPU*/
  int *d_batch_M;
  int *d_irregular_batch_M;
  hipMalloc(&d_batch_M,1024*sizeof(int));
  hipMalloc(&d_irregular_batch_M,h_irregular_batch_size*sizeof(int));
  

  /*Declaration et creation des streams*/
  hipStream_t stream[h_number_of_batches+h_is_irregular_batch];
  for (int ind_stream=0; ind_stream<h_number_of_batches; ind_stream++)
      hipStreamCreate(&stream[ind_stream]);
  if (h_is_irregular_batch==1)
      hipStreamCreate(&stream[h_number_of_batches]);


  /*Timer*/
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /*Slices inferieures a 1024*/
  hipEventRecord(start);
  while (h_number_of_slices > 0)
  {   
      /*Mise a jour taille et indices*/
      h_slice_size=2*h_slice_size;
      h_slice_reste_precedent=h_slice_reste;
      h_slice_reste=1024%h_slice_size;
      h_number_of_slices=1024/h_slice_size;
      
      for (int b=0; b<h_number_of_batches;b++)
      {   
          hipMemcpyAsync(d_batch_M, h_batch_M[b], 1024*sizeof(int), hipMemcpyHostToDevice, stream[b]);
          
          if (h_slice_reste_precedent!=0 && h_slice_reste!=0)
          {
              int h_taille_A_extra=h_slice_reste-h_slice_reste_precedent;
              int h_taille_B_extra=h_slice_reste_precedent;
              small_sortManager_extraSlice<<<h_number_of_slices+1,1024,0,stream[b]>>>(d_batch_M,h_slice_size/2,h_slice_size/2,h_taille_A_extra,h_taille_B_extra,1024,h_number_of_slices);
          }
          else
            small_sortManager<<<h_number_of_slices,1024,0,stream[b]>>>(d_batch_M, h_slice_size/2, h_slice_size/2,h_slice_size,h_number_of_slices);

          hipMemcpyAsync(h_batch_M[b], d_batch_M, 1024*sizeof(int), hipMemcpyDeviceToHost, stream[b]);
      }
      if (h_is_irregular_batch==1 && h_irregular_number_of_slices>0)
      {
          
           h_irregular_slice_size=2*h_irregular_slice_size;
           h_irregular_slice_reste_precedent=h_irregular_slice_reste;
           h_irregular_slice_reste=h_irregular_batch_size%h_irregular_slice_size;
           h_irregular_number_of_slices=h_irregular_batch_size/h_irregular_slice_size;

           hipMemcpyAsync(d_irregular_batch_M, h_irregular_batch_M, h_irregular_batch_size*sizeof(int), hipMemcpyHostToDevice, stream[h_number_of_batches]);

           if (h_irregular_slice_reste_precedent!=0 && h_irregular_slice_reste!=0)
           {
              
              int h_taille_A_extra=h_irregular_slice_reste-h_irregular_slice_reste_precedent;
              int h_taille_B_extra=h_irregular_slice_reste_precedent;
              small_sortManager_extraSlice<<<h_irregular_number_of_slices+1,h_irregular_batch_size,0,stream[h_number_of_batches]>>>(d_irregular_batch_M,h_irregular_slice_size/2,h_irregular_slice_size/2,h_taille_A_extra,h_taille_B_extra,h_irregular_batch_size,h_irregular_number_of_slices);

           }
           else
            small_sortManager<<<h_irregular_number_of_slices,h_irregular_batch_size,0,stream[h_number_of_batches]>>>(d_irregular_batch_M, h_irregular_slice_size/2, h_irregular_slice_size/2,h_irregular_slice_size,h_irregular_number_of_slices);
          
          hipMemcpyAsync(h_irregular_batch_M, d_irregular_batch_M, h_irregular_batch_size*sizeof(int), hipMemcpyDeviceToHost, stream[h_number_of_batches]);
          
      
      }
      

  }
  hipDeviceSynchronize();

  /*re ecriture de M*/
  for (int b=0; b<h_number_of_batches; b++)
      for (int ind=0; ind<1024; ind++)
          h_M[b*1024+ind]=h_batch_M[b][ind];
  if (h_is_irregular_batch==1)
      for (int ind=0; ind<h_irregular_batch_size; ind++)
          h_M[h_number_of_batches*1024+ind]=h_irregular_batch_M[ind];
  
  
  /*Slices superieures a 1024*/

  /*Mise a jour taille et indices*/
  h_slice_size=1024;


  /*Destruction des streams qui ne servent pas*/
  for (int i=(h_taille_M/h_slice_size); i<h_number_of_batches+h_is_irregular_batch; i++)
      hipStreamDestroy(stream[i]);
       

  /*Mise a jour taille et indices suite*/    
  h_number_of_slices=h_taille_M/h_slice_size;
  h_slice_reste=h_irregular_batch_size;
  int compteur=0;
  while (h_number_of_slices>0)
  {   
      compteur=compteur+1;
      /*Mise a jour taille et indices*/
      h_slice_size=2*h_slice_size;
   
      /*Destruction des streams qui ne servent pas*/
      if (compteur>1)
        for (int i=(h_taille_M/h_slice_size)+1; i<h_number_of_slices+1; i++)
          hipStreamDestroy(stream[i]);
   
      /*Mise a jour taille et indices suite*/
      h_slice_reste_precedent=h_slice_reste;
      h_slice_reste=h_taille_M%h_slice_size;
      h_number_of_slices=h_taille_M/h_slice_size;
      
      
      for (int i=0; i<h_number_of_slices; i++)
      {   
          sortManager_CPU(h_M,h_slice_size/2,h_slice_size/2,h_slice_size,i, stream);
          
      }
      if (h_slice_reste_precedent!=0 && h_slice_reste!=0)
      {
          int h_taille_A=h_slice_reste-h_slice_reste_precedent;
          int h_taille_B=h_slice_reste_precedent;
          sortManager_CPU(h_M,h_taille_A,h_taille_B,h_slice_size,h_number_of_slices,stream);

      }
      hipDeviceSynchronize();
       
  }
  
  hipDeviceSynchronize();
  hipEventRecord(stop);

  /*Affichage du chrono*/
  hipEventSynchronize(stop);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  fprintf(stderr,"mergeBatches Taille_M: %d, nbthreads: %d, numblocks: %d, Temps: %.5f, verif: %d\n", h_taille_M, threadsPerBlock, numBlocks, ms,verif_trie(h_M,h_taille_M));
  
  /*Destructions des streams restants*/
  for (int i=0; i<h_number_of_slices; i++)
    hipStreamDestroy(stream[i]);

  /*Verification*/
  if (verif_trie(h_M,h_taille_M)==-1)
    printf("ok tableau trie");
  else
    printf("KO recommencer %d ",verif_trie(h_M,h_taille_M) );

  /*Liberation*/
  hipFree(d_M);
  hipFree(d_batch_M);
  hipFree(d_irregular_batch_M);

  for (int b=0;b<h_number_of_batches;b++)
      free(h_batch_M[b]);
    
  free(h_M);
  free(h_batch_M);
  free(h_irregular_batch_M);


    return 0;
}