#include "hip/hip_runtime.h"
//*****************************************************************************
//Projet HPC fusion et trie de tableaux sur GPU
//Auteur: ROBIN Clement et SAULNIER Solene
//Promo: MAIN5
//Date: decembre 2020
//Question 5 mais seulement pour des batches de moins de 1024
//*****************************************************************************


#include <stdio.h> 
#include <stdlib.h>
#include <string.h>
#define N 1024
#define threadsPerBlock 1024
#define numBlock 65535

//*****************************************************************************
//Fonctions CPU verification
//*****************************************************************************
int verif_trie(int *tab,int size)
{
    for (int i=0; i<size-1; i=i+1)
      if (tab[i]>tab[i+1])
          return i;
    return 1;
    
}

//*****************************************************************************
//Fonctions GPU (merge tableau)
//*****************************************************************************

__device__ void mergeSmallBatch_k(int *A, int *B, int *M, int size_A, int size_B, int size_M, int slice_size)
{

    for(int i = threadIdx.x; i<size_A+size_B; i = i+blockDim.x*gridDim.x)
    {  
     
      /*Merge batches*/
      int K[2],P[2],Q[2];
      int offset;

      if (i>size_A)
      {
          K[0]=i-size_A;
          K[1]=size_A;
          P[0]=size_A;
          P[1]=i-size_A;
      }
      else
      {
          K[0]=0;
          K[1]=i;
          P[0]=i;
          P[1]=0;
          
      }
      while (1)
      {
          offset=abs(K[1]-P[1])/2;
          Q[1]=K[1]-offset;
          Q[0]=K[0]+offset;
          if (Q[1] >= 0 && Q[0] <= size_B && (Q[1]== size_A || Q[0]==0 || A[Q[1]]>B[Q[0]-1]))//verif
          {
              if (Q[0]==size_B || Q[1]==0 || A[Q[1]-1]<=B[Q[0]])//verif
              {
                  if (Q[1]<size_A && (Q[0]==size_B || A[Q[1]]<=B[Q[0]]))
                      M[blockIdx.x * slice_size + i]=A[Q[1]];
                  else
                      M[blockIdx.x * slice_size + i]=B[Q[0]]; 
                  break;
              }
              else
              {
                  K[0]=Q[0]+1;
                  K[1]=Q[1]-1;
              }
          }
          else
          {
            P[0]=Q[0]-1;
            P[1]=Q[1]+1;
          }
      }
    }
  
}

__global__ void sortManager(int *M, int size_A, int size_B, int size_M, int number_of_slices)
{

      int slice_size = size_A + size_B;
        

      /*Chargement de A et B dans la shared memory*/
      /*Comme on a une seule shared memory*/
      __shared__ int shared_AB[1024];  //Comme A et B ne peuvent pas depasser 1024
 
      int* s_A = (int*) &shared_AB[0];
      int* s_B = (int*) &s_A[size_A];
 
      __syncthreads();
 
      if (threadIdx.x < size_A)
        s_A[threadIdx.x] = M[blockIdx.x *slice_size+ threadIdx.x];
        
 
      if (threadIdx.x >= size_A && threadIdx.x < size_B + size_A  )
        s_B[threadIdx.x-size_A] = M[blockIdx.x *slice_size+ threadIdx.x]; 

      __syncthreads();

      mergeSmallBatch_k(s_A, s_B, M, size_A, size_B, size_M,slice_size); 
  
}

__global__ void sortManager_extraSlice(int *M ,int size_A,int size_B,int size_A_extra,int size_B_extra,int size_M,int number_of_slices)
{

      int slice_size = size_A + size_B;

      /*Chargement de A et B dans la shared memory*/
      /*Comme on a une seule shared memory*/
      __shared__ int shared_AB[1024];  //Comme A et B ne peuvent pas depasser 1024
 
      int* s_A = (int*) &shared_AB[0];
     
      int* s_B;
     
      if (blockIdx.x == number_of_slices)
        s_B = (int*) &s_A[size_A_extra];

      else
        s_B = (int*) &s_A[size_A];
 
      __syncthreads();

      if (threadIdx.x < size_A)
        s_A[threadIdx.x] = M[blockIdx.x *slice_size+ threadIdx.x]; 
 
      if (threadIdx.x >= size_A && threadIdx.x < size_B + size_A)
        s_B[threadIdx.x-size_A] = M[blockIdx.x *slice_size+ threadIdx.x]; 

      if (blockIdx.x == number_of_slices && threadIdx.x < size_A_extra)
        s_A[threadIdx.x] = M[blockIdx.x *slice_size+ threadIdx.x];
     
      if (blockIdx.x == number_of_slices && threadIdx.x >= size_A_extra && threadIdx.x < size_B_extra + size_A_extra)
        s_B[threadIdx.x-size_A_extra] = M[blockIdx.x *slice_size+ threadIdx.x]; 

      __syncthreads();

     if (blockIdx.x == number_of_slices)
     {
        if (size_A_extra < size_B_extra)
          mergeSmallBatch_k(s_B, s_A, M, size_B_extra, size_A_extra, size_M, slice_size);  
      
        else
          mergeSmallBatch_k(s_A, s_B, M, size_A_extra, size_B_extra, size_M, slice_size); 
     }

    else
      mergeSmallBatch_k(s_A, s_B, M, size_A, size_B, size_M,slice_size); 
    
 
}

 
//*****************************************************************************
//MAIN
//*****************************************************************************
int main(int argc, char *argv[]) {
    
  srand (time (NULL));

  /*Declaration des variables CPU*/
  /*Taille des tableaux*/
  int h_taille_M=N; 

  /*Tableaux et allocation memoire*/
  int *h_M;
  h_M=(int *)malloc(h_taille_M*sizeof(int));

  /*Declaration des variables GPU*/ 
  int *d_M;  
  hipMalloc(&d_M,h_taille_M*sizeof(int));

  /*Initialisation et preparation des tableaux*/
  for (int i=0; i<h_taille_M;i++)
    h_M[i]=rand()%10000;


  /*Transfert la memoire du cpu vers le gpu*/
  hipMemcpy(d_M, h_M, h_taille_M*sizeof(int), hipMemcpyHostToDevice);


  /*Timer*/
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /*Merge*/
  int h_slice_size=1;
  int h_number_of_slices=h_taille_M/2;
  int h_slice_reste_precedent=0;
  int h_slice_reste=0;


  hipEventRecord(start);
  while (h_number_of_slices > 0)
  {   
      /*Mise a jour taille et indices*/
      h_slice_size=2*h_slice_size;
      h_slice_reste_precedent=h_slice_reste;
      h_slice_reste=h_taille_M%h_slice_size;
      h_number_of_slices=h_taille_M/h_slice_size;
      
   
      if (h_slice_reste_precedent!=0 && h_slice_reste!=0)
      {
          int h_taille_A_extra=h_slice_reste-h_slice_reste_precedent;
          int h_taille_B_extra=h_slice_reste_precedent;
          sortManager_extraSlice<<<h_number_of_slices+1,h_taille_M>>>(d_M,h_slice_size/2,h_slice_size/2,h_taille_A_extra,h_taille_B_extra,h_taille_M,h_number_of_slices);
      }
      else
        sortManager<<<h_number_of_slices,h_taille_M>>>(d_M, h_slice_size/2, h_slice_size/2,h_slice_size,h_number_of_slices);

  }
  hipDeviceSynchronize();
  hipEventRecord(stop);



  /*Transfert la memoire du gpu vers le cpu*/
  hipMemcpy(h_M, d_M, h_taille_M*sizeof(int), hipMemcpyDeviceToHost);


  /*Affichage du chrono*/
  hipEventSynchronize(stop);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  fprintf(stderr,"mergeSmallBatch_Only Taille_M: %d, nbthreads: %d, numblocks: %d, Temps: %.5f, verif: %d\n", h_taille_M,threadsPerBlock,numBlock,ms,verif_trie(h_M,h_taille_M));
  


  /*Verification*/
  if (verif_trie(h_M,h_taille_M)==1)
    printf("\n ok tableau trie\n");
  else
    printf("\n KO probleme a l indice %d\n",verif_trie(h_M,h_taille_M));
   
  /*Liberation*/
  hipFree(d_M);
  free(h_M);
 

    return 0;
}