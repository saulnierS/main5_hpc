//*****************************************************************************
//Projet HPC fusion et trie de tableaux sur GPU
//Auteur: ROBIN Clement et SAULNIER Solene
//Promo: MAIN5
//Date: decembre 2020
//Question 1
//*****************************************************************************


#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>
#include <string.h>
#define N 1024
#define threadsPerBlock 1024
//*****************************************************************************
//Fonctions CPU fusion et verification
//*****************************************************************************
int verif_trie(int *tab,int size)
{
    for (int i=0; i<size-1; i=i+1)
      if (tab[i]>tab[i+1])
          return i;
    return 1;
    
}

void fusion(int* tableau,int deb1,int fin1,int fin2)
{
  int *table1;
  int deb2=fin1+1;
  int compt1=deb1;
  int compt2=deb2;
  int i;
        
  table1=(int *) malloc((fin1-deb1+1)*sizeof(int));

  //on recopie les elements du debut du tableau
  for(i=deb1;i<=fin1;i++)
  {
      table1[i-deb1]=tableau[i];
  }
                  
  for(i=deb1;i<=fin2;i++)
  {        
    if (compt1==deb2) //c'est que tous les elements du premier tableau ont ete utilises
    {
      break; //tous les elements ont donc ete classes
    }
    else if (compt2==(fin2+1)) //c'est que tous les elements du second tableau ont ete utilises
    {
      tableau[i]=table1[compt1-deb1]; //on ajoute les elements restants du premier tableau
      compt1++;
    }
    else if (table1[compt1-deb1]<tableau[compt2])
    {
      tableau[i]=table1[compt1-deb1]; //on ajoute un element du premier tableau
      compt1++;
    }
    else
    {
      tableau[i]=tableau[compt2]; //on ajoute un element du second tableau
      compt2++;
    }
  }
  free(table1);
}
        

void tri_fusion_bis(int* tableau,int deb,int fin)
{
  if (deb!=fin)
  {
    int milieu=(fin+deb)/2;
    tri_fusion_bis(tableau,deb,milieu);
    tri_fusion_bis(tableau,milieu+1,fin);
    fusion(tableau,deb,milieu,fin);
  }
}

void tri_fusion(int* tableau,int longueur)
{
  if (longueur>0)
  {
    tri_fusion_bis(tableau,0,longueur-1);
  }
}
//*****************************************************************************
//Fonctions GPU (merge tableau)
//*****************************************************************************
__global__ void mergeSmall_k(int *A, int *B, int *M, int size_A, int size_B, int size_M)
{
    
    for( int i = blockIdx.x * blockDim.x  + threadIdx.x; i< size_M; i = i+ blockDim.x*gridDim.x)
    {
        
      int K[2],P[2],Q[2];
      int offset;

      
      if (i>size_A)
      {
          K[0]=i-size_A;
          K[1]=size_A;
          P[0]=size_A;
          P[1]=i-size_A;
      }
      else
      {
          K[0]=0;
          K[1]=i;
          P[0]=i;
          P[1]=0;
      }
      while (1)
      {
          offset=abs(K[1]-P[1])/2;
          Q[1]=K[1]-offset;
          Q[0]=K[0]+offset;
          if (Q[1] >= 0 && Q[0] <= size_B && (Q[1]== size_A || Q[0]==0 || A[Q[1]]>B[Q[0]-1]))
          {
              if (Q[0]==size_B || Q[1]==0 || A[Q[1]-1]<=B[Q[0]])
              {
                  if (Q[1]<size_A && (Q[0]==size_B || A[Q[1]]<=B[Q[0]]))
                  {
                      M[i]=A[Q[1]];
                  }
                  else
                  {
                      M[i]=B[Q[0]]; 
                  }
                  break;
              }
              else
              {
                  K[0]=Q[0]+1;
                  K[1]=Q[1]-1;
              }
          }
          else
          {
            P[0]=Q[0]-1;
            P[1]=Q[1]+1;
          }
      }
    }
}


 
//*****************************************************************************
//MAIN
//*****************************************************************************
int main(int argc, char *argv[]) {

  srand(42);
  int numThreads=threadsPerBlock;

  /*Declaration des variables CPU*/
  /*Taille des tableaux*/
  int h_taille_A=N-10;
  int h_taille_B=N-h_taille_A;
  int h_taille_M=h_taille_A+h_taille_B;

  for (int i=0; i<argc-1; i=i+1)
  {
      if (strcmp(argv[i],"--tailleA")==0 && atoi(argv[i+1])<N )
          h_taille_A=atoi(argv[i+1]);
      if (strcmp(argv[i],"--tailleB")==0 && atoi(argv[i+1])<N)
          h_taille_B=atoi(argv[i+1]);
      if (strcmp(argv[i],"--nbthreads")==0 && atoi(argv[i+1])<threadsPerBlock )
          numThreads=atoi(argv[i+1]);     
  }

  if (h_taille_A < h_taille_B)
  {
      int tpm=h_taille_A;
      h_taille_A=h_taille_B;
      h_taille_B=tpm;
  }

  /*Tableaux et allocation memoire*/
  int *h_A;
  int *h_B;
  int *h_M;
  h_A=(int *)malloc(h_taille_A*sizeof(int));
  h_B=(int *)malloc(h_taille_B*sizeof(int));
  h_M=(int *)malloc(h_taille_M*sizeof(int));
 

  /*Declaration des variables GPU*/ 
  int *d_A; 
  int *d_B; 
  int *d_M;
  hipMalloc(&d_A,h_taille_A*sizeof(int)); 
  hipMalloc(&d_B,h_taille_B*sizeof(int));
  hipMalloc(&d_M,h_taille_M*sizeof(int));
  
   
  /*Initialisation et preparation des tableaux*/
  for (int i=0; i<h_taille_A;i++)
    h_A[i]=rand()%10000;
  
  for (int i=0; i<h_taille_B;i++)
    h_B[i]=rand()%10000;
  
  tri_fusion(h_A, h_taille_A);
  tri_fusion(h_B, h_taille_B);


  /*Transfert la memoire du cpu vers le gpu*/
  hipMemcpy(d_A, h_A, h_taille_A*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, h_taille_B*sizeof(int), hipMemcpyHostToDevice);

  /*Timer*/
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /*Merge tableau*/
  hipEventRecord(start);
  mergeSmall_k<<<1,numThreads>>>(d_A, d_B, d_M, h_taille_A, h_taille_B, h_taille_M);
  hipDeviceSynchronize();
  hipEventRecord(stop);

 
  /*Transfert la memoire du gpu vers le cpu*/
  hipMemcpy(h_M, d_M, h_taille_M*sizeof(int), hipMemcpyDeviceToHost);
  
  /*Affichage du chrono*/
  hipEventSynchronize(stop);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  fprintf(stderr,"mergeSmall_k Taille_A: %d, Taille_B: %d, Taille_M: %d, nbthreads: %d, Temps: %.5f, verif: %d\n", h_taille_A, h_taille_B, h_taille_M,numThreads,ms,verif_trie(h_M,h_taille_M));
  
  
  /*Verification*/
  if (verif_trie(h_M,h_taille_M)==1)
    printf("\n ok tableau M trie\n");
  else
    printf("\n KO probleme a l indice %d\n",verif_trie(h_M,h_taille_M));


  /*Liberation*/
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_M);
  free(h_A);
  free(h_B);
  free(h_M);
 

    return 0;
} 