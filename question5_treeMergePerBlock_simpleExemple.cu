#include "hip/hip_runtime.h"
//%%writefile treeMergePerBlock_simpleExemple.cu
#include <stdio.h> 
#include <stdlib.h>
#include <string.h>
#include "fonctionsCPU.h"
#define N 100000
#define threadsPerBlock 1024
#define numBlock 65535
//*****************************************************************************
//Fonctions GPU (merge tableau)
//*****************************************************************************

__device__ void mergeSmallBatch_k(int *A, int *B, int *M, int size_A, int size_B, int size_M, int slice_size)
{

    int i = threadIdx.x;
    if (i < size_A+size_B)
    {
        //printf("\n blockIdx = %d, threadIdx = %d \n",blockIdx.x,threadIdx.x);
      
     
     /*Merge*/
      int K[2],P[2],Q[2];
      int offset;

      //if (i==0) printf("A: %d B:%d M:%d\n", size_A, size_B, size_M);
      if (i>size_A)
      {
          K[0]=i-size_A;
          K[1]=size_A;
          P[0]=size_A;
          P[1]=i-size_A;
      }
      else
      {
          K[0]=0;
          K[1]=i;
          P[0]=i;
          P[1]=0;
          //if (i==0) printf("K(%d,%d) et P(%d,%d)\n",K[0],K[1],P[0],P[1]);
      }
      while (1)
      {
          offset=abs(K[1]-P[1])/2;
          //if (i==0) printf("K(%d,%d) et P(%d,%d)\n",K[0],K[1],P[0],P[1]);
          Q[1]=K[1]-offset;
          Q[0]=K[0]+offset;
          if (Q[1] >= 0 && Q[0] <= size_B && (Q[1]== size_A || Q[0]==0 || A[Q[1]]>B[Q[0]-1]))//verif
          {
              if (Q[0]==size_B || Q[1]==0 || A[Q[1]-1]<=B[Q[0]])//verif
              {
                  if (Q[1]<size_A && (Q[0]==size_B || A[Q[1]]<=B[Q[0]]))//verif
                  {
                      M[blockIdx.x * slice_size + i]=A[Q[1]];
                   //printf("\n blockIdx = %d, threadIdx = %d : M[%d] = A[%d]\n",blockIdx.x,threadIdx.x,blockIdx.x * slice_size + i,Q[1]);
                  }
                  else
                  {
                      M[blockIdx.x * slice_size + i]=B[Q[0]]; 
                    //printf("\n blockIdx = %d, threadIdx = %d : M[%d] = B[%d]\n",blockIdx.x,threadIdx.x,blockIdx.x * slice_size + i,Q[0]);
                  }
                  break;
              }
              else
              {
                  K[0]=Q[0]+1;
                  K[1]=Q[1]-1;
              }
          }
          else
          {
            P[0]=Q[0]-1;
            P[1]=Q[1]+1;
          }
      }
    }
  //printf("pour %d on fait %d tours \n",blockIdx.x * blockDim.x + threadIdx.x,c);
}

__global__ void sortManager(int *M, int size_A, int size_B, int size_M)
{
    
        int slice_size = size_A + size_B;
     
        int number_of_slices = size_M/slice_size;
        

          
  

       /*Chargement de A et B dans la shared memory*/
      /*Comme on a une seule shared memory*/
      __shared__ int shared_AB[1024];  //Comme A et B ne peuvent pas dépasser 1024
 
      int* s_A = (int*) &shared_AB[0];
      int* s_B = (int*) &s_A[size_A];
 
      __syncthreads();
 
      //int i = blockDim.x * blockIdx.x + threadIdx.x;
      if (threadIdx.x < size_A)
      {
          s_A[threadIdx.x] = M[blockIdx.x *slice_size+ threadIdx.x];
          //printf("\n blockIdx = %d, threadIdx = %d : s_A[%d] = M[%d] = %d\n",blockIdx.x,threadIdx.x,threadIdx.x, blockIdx.x *slice_size+ threadIdx.x, M[blockIdx.x *slice_size+ threadIdx.x]);
      }
        
 
      if (threadIdx.x >= size_A && threadIdx.x < size_B + size_A  )
      {
          s_B[threadIdx.x-size_A] = M[blockIdx.x *slice_size+ threadIdx.x]; 
          //printf("\n blockIdx = %d, threadIdx = %d : s_B[%d] = M[%d] = %d\n",blockIdx.x,threadIdx.x,threadIdx.x, blockIdx.x *slice_size+ threadIdx.x, M[blockIdx.x *slice_size+ threadIdx.x]);
      }
        

      __syncthreads();

      mergeSmallBatch_k(s_A, s_B, M, size_A, size_B, size_M,slice_size); 
  
 
   
}

 
//*****************************************************************************
//MAIN
//*****************************************************************************
int main(int argc, char *argv[]) {
    
  srand (time (NULL));
  /*Déclaration des variables CPU*/
  /*Taille des tableaux*/

  int h_taille_M=8;
  
  printf("taille de M:%d \n",h_taille_M);

  /*Tableaux et allocation memoire*/
  int *h_M;
  h_M=(int *)malloc(h_taille_M*sizeof(int));
 

  /*Déclaration des variables GPU*/ 
  int *d_M;
  hipMalloc(&d_M,h_taille_M*sizeof(int));
    

   
  /*Initialisation et preparation des tableaux*/

  h_M[0] = 8;
  h_M[1] = 4;
  h_M[2] = 12;
  h_M[3] = 2;
  h_M[4] = 2;
  h_M[5] = 1;
  h_M[6] = 5;
  h_M[7] = 7;



  /*Affichage*/

     printf("\n");
  printf("***M***\n");
  for (int i=0; i<h_taille_M; i=i+1)
    printf("%d\n",h_M[i]);

  /*Transfert la mémoire du cpu vers le gpu*/
  hipMemcpy(d_M, h_M, h_taille_M*sizeof(int), hipMemcpyHostToDevice);

  int h_slice_size = 2;
  int h_number_of_slices = h_taille_M / h_slice_size;
  /*Merge tableau*/
  sortManager<<<h_number_of_slices,8>>>(d_M, 1, 1, h_taille_M);


  hipMemcpy(h_M, d_M, h_taille_M*sizeof(int), hipMemcpyDeviceToHost);

  printf("\n");
  printf("***M*** avec slice_size = %d\n",h_slice_size);
  for (int i=0; i<h_taille_M; i=i+1)
    printf("%d\n",h_M[i]);


  h_slice_size = 4;
  h_number_of_slices = h_taille_M / h_slice_size;
  sortManager<<<h_number_of_slices,8>>>(d_M, h_slice_size/2, h_slice_size/2, h_taille_M);


    printf("\n");
  printf("***M*** avec slice_size = %d\n",h_slice_size);
  for (int i=0; i<h_taille_M; i=i+1)
    printf("%d\n",h_M[i]);


  h_slice_size = 8;
  h_number_of_slices = h_taille_M / h_slice_size;
  sortManager<<<h_number_of_slices,8>>>(d_M, h_slice_size/2, h_slice_size/2, h_taille_M);

  /*Transfert la mémoire du gpu vers le cpu*/
  hipMemcpy(h_M, d_M, h_taille_M*sizeof(int), hipMemcpyDeviceToHost);

  /*Affichage du resultat*/
  
  printf("\n");
  printf("***M*** avec slice_size = %d\n",h_slice_size);
  for (int i=0; i<h_taille_M; i=i+1)
    printf("%d\n",h_M[i]);
  
  if (verif_trie(h_M,h_taille_M)==1)
    printf("\n ok tableau trié\n");
    else
  {
    printf("\n KO probleme a l indice %d\n",verif_trie(h_M,h_taille_M));
    //printf("%d %d %d %d\n",h_M[verif_trie(h_M,h_taille_M)-1],h_M[verif_trie(h_M,h_taille_M)], h_M[verif_trie(h_M,h_taille_M)+1], h_M[verif_trie(h_M,h_taille_M)+2] );
  }
 
  hipFree(d_M);
  free(h_M);
 

    return 0;
}
    